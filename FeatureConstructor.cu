#include "hip/hip_runtime.h"
#include "FeatureConstructor.h"
#include "Cfg.h"

using namespace std;

#undef get16bits
#if (defined(__GNUC__) && defined(__i386__)) || defined(__WATCOMC__) \
|| defined(_MSC_VER) || defined (__BORLANDC__) || defined (__TURBOC__)
#define get16bits(d) (*((const uint16_t *) (d)))
#endif

#if !defined (get16bits)
#define get16bits(d) ((((uint32_t)(((const uint8_t *)(d))[1])) << 8)\
+(uint32_t)(((const uint8_t *)(d))[0]) )
#endif

 

dataCollection dataCollection_new()
{
	 dataCollection a;

  return a;
}

FeatureConstructor::FeatureConstructor()
{
    
}

FeatureConstructor::FeatureConstructor(int* document_size, int number_documents)
{
	
    label_list= new string[MAX_NUM_LABELS];
    num_labels=0;
	num_unique_words=0;
}


int calculate_table_size(int* documents_size, int number_documents)
{
    int result=0;
    for(int i=0;i<number_documents;i++)
        result+=documents_size[i];
    return  result;
}

FeatureConstructor::FeatureConstructor(int* document_size, int number_documents,int total_characters_count,int total_words_count)
{
    num_labels=0;
    num_unique_words=0;
	documents_size = document_size;

    total_char_count=total_characters_count;
	total_words_count= total_words_count;
	vocab_list= new string[total_words_count];
    label_list= new string[MAX_NUM_LABELS];
    hash_list= new LinkedList[HASH_TABLE_SIZE];
    max_List_Size=0;
    for(int i=0;i<HASH_TABLE_SIZE;i++)
    {
        hash_list[i]= *new LinkedList();
    }
}

bool FeatureConstructor::check_if_feature(string word)
{
    bool check = true;
    
    if(word.length() <= 3)
    {
        check = false;
    }
    else if(word == "though" || word == "they" || word == "that" || word == "this" || word == "there" || word == "were"
            || word == "than" || word == "rather" || word == "from" || word == "most")
        check = false;
    
    return check;
}

int FeatureConstructor::hash_str(string word, int tableSize)
{
    int h = 31 /* also prime */;
    
    char * charList= (char *)malloc(word.length()*sizeof(char));
    for(int i=0;i<word.length();i++)
        charList[i]= word[i];
    
    
    while (*charList) {
       // h = (h * A) ^ (charList[0] * B);
        charList++;
    }
    return h%tableSize; // or return h % C;
}

void FeatureConstructor::extract_labels(string** data_list, int number_documents)
{
	for(int i=0;i<number_documents;i++)
    {
		bool found=false;
        // loop over all discovered labes see if it already added or not
        for(int l=0;l<num_labels; l++)
        {
            if (data_list[i][0] == label_list[l])
            {
                // if found, set the flag to true and replace the lablel with its index in the labellist
                found= true;
            }
        }
        
        // if the label isn't found in the label list
        if(!found && data_list[i][0] != "")
        {
            string s = data_list[i][0];
            printf("Index: %d\tLabel: %s\tDoc: %d\n",num_labels,data_list[i][0].c_str(),i);
            // add it to the list
            label_list[num_labels]= data_list[i][0];
            
            // increment the number of labels
            num_labels++;
            
        }
    }
    printf("Labels found: %d\n",num_labels);
}

void FeatureConstructor::extract_documents_indexes(int* documents_size, int number_documents)
{
	documents_indexes = (int*)malloc(sizeof(int)*number_documents);

	for(int i = 0;i < number_documents;i++)
	{
		documents_indexes[i] = get_document_index(documents_size,i);
	}
}

//Builds a list of all unique words in vocab_list from data_list
void FeatureConstructor::extract_vocab(string** data_list,int* documents_size, int number_documents)
{
    
    // loop over all documents and extract all labels
    printf("Began Vocab Extraction\n");
    
	extract_labels(data_list,number_documents);

    // loop over documents
    for(int i=0;i<number_documents; i++)
    {
        if( ((float)i/number_documents) == ((float)80/100) )
            printf("80%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)60/100))
            printf("60%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)40/100))
            printf("40%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)20/100))
            printf("20%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)10/100))
            printf("10%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(i == number_documents - 1)
            printf("100%% Completed: %d Unique words found: %d\n",i,num_unique_words);
            
        // loop over all words in this document
        for(int j=1;j<documents_size[i];j++)
        {
            if(!check_if_feature(data_list[i][j]))
                continue;
            
            // calculate the hash index
            int hashIndex= SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
            // if the word wasn't already added to the list in the specified index
            if(!hash_list[hashIndex].Exists(data_list[i][j]))
            {
                // append word to the list
                hash_list[hashIndex].Append((data_list[i][j]),num_unique_words);
				// update the vocablist
				vocab_list[num_unique_words]=data_list[i][j];
				//printf("%s ",data_list[i][j]);
                num_unique_words++;
            }
           
        }
    }

	//printf("\n");
    
    printf("Ended Vocab Extraction\n");
}

int FeatureConstructor::get_index_for_label(string label)
{
    int index = -1;
    
    for(int i = 0;i < num_labels;i++)
    {
        if(label == label_list[i])
        {
            index = i;
            break;
        }
    }
    
    return index;
}

void FeatureConstructor::convert_labels_integers(string ** data_list, int number_documents)
{
    for(int i = 0;i<number_documents;i++)
    {
        documents_labels[i] = get_index_for_label(data_list[i][0]);
    }
}

int FeatureConstructor::get_document_index(int* documents_size, int doc_num)
{
	int index = 0;

	for(int i = 0;i<doc_num;i++)
	{
		index += documents_size[i];
	}

	return index;
}

//Builds feature vectors for all documents in document_feature_vectors from data_list
void FeatureConstructor::construct_feature_vectors(string** data_list,int* documents_size, int number_documents)
{
    printf("Began Feature Construction\n");
    int hashIndex, position;
	int totalSize = calculate_table_size(documents_size,number_documents);

    // set the number of rows to be equal number of documents
    feature_vector =  (int*)malloc(sizeof(int)*totalSize);
    documents_labels = (int*)malloc(sizeof(int)*number_documents);

    convert_labels_integers(data_list, number_documents);
	extract_documents_indexes(documents_size, number_documents);
    
    // loop on every row and set number of columns to be equal of number of unique words
    for(int i=0;i<totalSize;i++)
    {
		feature_vector[i]=0;
    }

    // loop over every document
    for(int i=0;i<number_documents;i++)
    {
        for(int j=0;j<documents_size[i];j++)
        {
            // check if word can be used as feature word
            if(!check_if_feature(data_list[i][j]))
			{
				position = -1;
			}
			else
			{
				// calculate index
				hashIndex = SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
				// get the position of the word int he vocablist
				position = hash_list[hashIndex].getPositionValue(data_list[i][j]);
			}

			feature_vector[documents_indexes[i] + j] = position;
            
			//printf("%i ",feature_vector[get_document_index(documents_size,i) + j]);
        }
		//printf("\n");
		//printf("\n");
        
    }

	
	/***** DEALLOCATE UNUSED ARRAYS *****/
	//for(int i = 0;i<HASH_TABLE_SIZE;i++)
	//{
	//	hash_list[i].Destroy();
	//}

	//delete[] hash_list;

    printf("Ended Feature Construction\n");
    
}

__device__ uint32_t hash_inc(const char * data, int len, uint32_t hash)
{
    uint32_t tmp;
    int rem;
    
    if (len <= 0 || data == NULL) return 0;
    
    rem = len & 3;
    len >>= 2;
    
    /* Main loop */
    for (;len > 0; len--) {
        hash  += get16bits (data);
        tmp    = (get16bits (data+2) << 11) ^ hash;
        hash   = (hash << 16) ^ tmp;
        data  += 2*sizeof (uint16_t);
        hash  += hash >> 11;
    }
    
    /* Handle end cases */
    switch (rem) {
        case 3: hash += get16bits (data);
            hash ^= hash << 16;
            hash ^= data[sizeof (uint16_t)] << 18;
            hash += hash >> 11;
            break;
        case 2: hash += get16bits (data);
            hash ^= hash << 11;
            hash += hash >> 17;
            break;
        case 1: hash += *data;
            hash ^= hash << 10;
            hash += hash >> 1;
    }
    
    /* Force "avalanching" of final 127 bits */
    hash ^= hash << 3;
    hash += hash >> 5;
    hash ^= hash << 4;
    hash += hash >> 17;
    hash ^= hash << 25;
    hash += hash >> 6;
    
	return hash% HASH_TABLE_SIZE;
} 

__device__ int compare(char* a, char* b, int length_a)
{
  for(int i=0;i<length_a;i++)
  {
	  if(a[i]!=b[i])
		return 0;
  }
	  if(b[length_a]=='&')
		 return 1;
	  else
	  return 0;
}

__device__ int atomic_add(int *address, int value)
{
	int result=atomicAdd(address,value);
	return result;
}

__global__ void extract_vocab_kernel(dataCollection d_data  )
{
	int threadId = threadIdx.x + blockIdx.x*blockDim.x;
	char *word;
	int startIndex, wordLength, hashValue;
	bool wordFound=false;

	
	
		// get the start index of the current word
	startIndex=d_data.d_indexes_string_start[threadId];
	word= &d_data.d_charDataList[startIndex];
		//check if the current word is not the last word
	if((threadId+1)< d_data.total_word_count)
	{
		// get the word length = start of next word - start of current word
		wordLength= d_data.d_indexes_string_start[threadId+1]- d_data.d_indexes_string_start[threadId];
	}
	else
	{
		// case the word is the last word:get word length = total number of chars - start of current word
		wordLength= d_data.total_char_count- d_data.d_indexes_string_start[threadId];
	}


	// get the hash value of the current word
	hashValue= hash_inc(word,wordLength,(uint32_t) wordLength);
	
	//int oldNumOfWords1=  atomic_add(&d_data.d_words_per_hash_row[hashValue],1);

	for(int j=0;j< d_data.d_words_per_hash_row[hashValue];j++)
	{
		int wordStartinHashTable=hashValue*d_data.hash_row_size*d_data.hash_word_size+ j*d_data.hash_word_size; 
		if(compare(&d_data.d_charDataList[startIndex],&d_data.d_HashArray[wordStartinHashTable], wordLength))
		{
			wordFound= true;
			break;
		}
	}

	
	//int oldNumOfWords1=  atomic_add(&d_data.d_words_per_hash_row[hashValue],1);
	if(!wordFound)
	{
		// update the number of words in this hash row
		int oldNumOfWords=  atomic_add(&d_data.d_words_per_hash_row[hashValue],1);
		// caluclat the  new index where the word will be written into 
		int hashArrayIndex=hashValue*d_data.hash_row_size*d_data.hash_word_size+ oldNumOfWords*d_data.hash_word_size;
		//int hashArrayIndex=0;
		// copy data from data list to hash value
		for(int k=0;k<wordLength;k++)
		{
			d_data.d_HashArray[hashArrayIndex+k]= d_data.d_charDataList[startIndex+k];
		}
		d_data.d_HashArray[hashArrayIndex+wordLength]='&';
		//hipMemcpy(&d_data.d_HashArray[hashArrayIndex],&d_data.d_charDataList[startIndex],wordLength*sizeof(char),hipMemcpyDeviceToDevice);
	}


	

}

dataCollection FeatureConstructor::extract_vocab(string** data_list,int* documents_size, int number_documents, int total_word_count, int total_char_count)
{

     //loop over all documents and extract all labels
	
    printf("Began Labels Extraction\n");

	extract_labels(data_list,number_documents);
	
    printf("End of  Labels Extraction\n");

    printf("Began Vocab Extraction\n");

	   //convert the datalist to char list and extract both indexes
    //convert 2D array to 1D array of strings
	string* h_dl = CudaStd::convert_2d_to_1d_string(data_list, number_documents,documents_size,total_word_count);
	// get start index of each word
    int* indexes_string_start_data=(int*)malloc(sizeof(int)*total_word_count);//indexes for the start and length of chars
    //convert 1D array of strings to 1D array of chars
	char* h_charDataList = CudaStd::convert_string_arr_to_char_arr(h_dl,total_word_count,total_char_count,indexes_string_start_data);

	char* h_hash_array=(char*) malloc(sizeof(char)*HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE);
	int* h_words_per_hash_row= (int*)malloc(sizeof(int)*HASH_TABLE_SIZE);
	for(int i=0;i<HASH_TABLE_SIZE;i++)
	{
		h_words_per_hash_row[i]=0;
	}
	


	char *d_hash_array, *d_charDataList;
	int *d_words_per_hash_row, *d_indexes_string_start_data  ;
	
	//allocate memory of th char data list
	hipMalloc((void**)&d_charDataList,sizeof(char)*total_char_count);
	hipMalloc((void**)&d_indexes_string_start_data,sizeof(int)*total_word_count);
	hipMalloc((void**)&d_hash_array,sizeof(char)*HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE);
	hipMalloc((void**)&d_words_per_hash_row,sizeof(int)*HASH_TABLE_SIZE);

	
	hipMemcpy(d_charDataList,h_charDataList,total_char_count*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_indexes_string_start_data,indexes_string_start_data,total_word_count*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_words_per_hash_row,h_words_per_hash_row,HASH_TABLE_SIZE*sizeof(int),hipMemcpyHostToDevice);
    
	
    const int THREADS_X = 512;
    const int THREADS_Y = 1;
    const int BLOCKS_X = ceil(total_word_count/(float)THREADS_X);
    const int BLOCKS_Y = 1;

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	dataCollection d_data;
	d_data.d_charDataList= d_charDataList;
	d_data.d_indexes_string_start= d_indexes_string_start_data;
	d_data.d_HashArray= d_hash_array;
	d_data.d_words_per_hash_row= d_words_per_hash_row;
	d_data.total_char_count= total_char_count;
	d_data.total_word_count= total_word_count;
	d_data.hash_table_size= HASH_TABLE_SIZE;
	d_data.hash_row_size= HASH_ROW_SIZE;
	d_data.hash_word_size= HASH_WORD_SIZE;

	extract_vocab_kernel<<<blocks,threads>>>(d_data);
	//hipMemcpy(h_hash_array,d_hash_array,HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE*sizeof(char),hipMemcpyDeviceToHost);
	hipMemcpy(h_words_per_hash_row,d_words_per_hash_row,sizeof(int)*HASH_TABLE_SIZE,hipMemcpyDeviceToHost);
 
	for(int i=0;i<HASH_TABLE_SIZE;i++)
	{
		num_unique_words+= h_words_per_hash_row[i];
	}

	free(h_charDataList);
	free(h_words_per_hash_row);
	free(h_hash_array);
	free(indexes_string_start_data);
	//delete(h_dl);

    printf("End of Vocab Extraction\n");

	return d_data;
}



#ifdef CUDA_FEATURE_VECTOR


__device__ int calculate_hash(char* word, int length)
{
	return 0;
}
/*__device__ int check_feature(char* word)
{
	bool check = true;
    if( compare(word,"though")==0 ||compare(word,"they")==0 || compare(word,"that")==0 || compare(word,"this")==0 || compare(word,"there")==0 || 
            compare(word,"were")==0 || compare(word,"than")==0 || compare(word,"rather")==0 || compare(word,"from")==0 || compare(word,"most")==0 )
        check = false;
    
    return check;
}*/
__global__ void construct_feature_vector_kernel(int* d_fv, char* d_dl, char* d_vl, int* start_index_data,int* d_table_indexes,int* start_index_vocab, int* d_positions)
{
	int threadId = threadIdx.x + blockIdx.x*blockDim.x;
	int my_word_start=start_index_data[threadId];
	char* my_word=&d_dl[my_word_start];
	int my_word_length=start_index_data[threadId+1]-start_index_data[threadId];
	int index_list=calculate_hash(my_word,my_word_length);
	int vocab_list_index=d_table_indexes[index_list];
	int vocab_list_length=d_table_indexes[index_list+1];

	for(int i=vocab_list_index;i<vocab_list_length;i=+start_index_vocab[i+1]-start_index_vocab[i])
	{
		char* vocab=&d_vl[i];
		int vocab_length=start_index_vocab[i+1]-start_index_vocab[i];
		if(compare(my_word,vocab,my_word_length)==1)
		{
			d_fv[threadId]=d_positions[i];
			break;
		}
	}
}
#endif

#ifdef CUDA_FEATURE_VECTOR
void FeatureConstructor::construct_feature_vectors(string** data_list,int* documents_size, int number_documents,int total_char_count,int total_word_count)
{
    printf("Began Feature Construction\n");
    int hashIndex, position;
	//int totalSize = calculate_table_size(documents_size,number_documents);

    // set the number of rows to be equal number of documents
    feature_vector =  (int*)malloc(sizeof(int)*total_word_count);
    documents_labels = (int*)malloc(sizeof(int)*number_documents);

    convert_labels_integers(data_list, number_documents);
	extract_documents_indexes(documents_size, number_documents);
    
    // loop on every row and set number of columns to be equal of number of unique words
    for(int i=0;i<total_word_count;i++)
    {
		feature_vector[i]=-1;
    }
	const int THREADS_Y = 1;
	const int THREADS_X = 512;
	const int BLOCKS_X = ceil(total_word_count/(float)THREADS_X);
	const int BLOCKS_Y = 1;

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int* d_fv;//feature vector array (output)
	char* d_cl;//char list array of the data list
	char* d_vl;//hash table array
	int* d_vl_index_start;//index of the d_vl
	int* d_table_indexes;//index of the lists in the hash table
	int* d_cl_index_start;//index of the d_cl
	int* d_positions;//positions to be put in FV

	//******TEMP FOR VOCAB COMPUTATION
	int vocab_list_total_char_count=0;
	for(int i=0;i<num_unique_words;i++)
	{
		vocab_list_total_char_count+= vocab_list[i].length();
	}
	//**************
	//************TEMP FOR HASHTABLE
	int* temp_positions;
	int* temp_hashLists;
	//**************


	//allocate feature vector, char list(data list->chars), vocab list->char,index for vocab list and data list
	hipMalloc((void**)&d_fv,total_word_count*sizeof(int));

	hipMalloc((void**)&d_cl,total_char_count*sizeof(char));
	hipMalloc((void**)&d_vl,vocab_list_total_char_count*sizeof(char));

	hipMalloc((void**)&d_vl_index_start,num_unique_words*sizeof(int)+sizeof(int));
	hipMalloc((void**)&d_table_indexes,num_unique_words*sizeof(int));//****TO DO get the size

	hipMalloc((void**)&d_cl_index_start,total_word_count*sizeof(int)+sizeof(int));
	hipMalloc((void**)&d_positions,num_unique_words*sizeof(int));


	//convert the datalist to char list and extract both indexes
	string* h_dl;//temp to hold the data into 1D string array
	//convert 2D array to 1D array of strings
	h_dl = CudaStd::convert_2d_to_1d_string(data_list, number_documents,documents_size,total_word_count);
	int* indexes_string_start_data=(int*)malloc(sizeof(int)*total_word_count);//indexes for the start and length of chars
	//int* indexes_string_length_data=(int*)malloc(sizeof(int)*total_word_count);
	//convert 1D array of strings to 1D array of chars
	char* h_cl = CudaStd::convert_string_arr_to_char_arr(h_dl,total_word_count,total_char_count,indexes_string_start_data);

	
	//processing the vocab list
	int* indexes_string_start_vocab=(int*)malloc(sizeof(int)*num_unique_words);//indexes for the start and length of chars
	//int* indexes_string_length_vocab=(int*)malloc(sizeof(int)*num_unique_words);
	//convert 1D array of strings to 1D array of chars
	char* h_vl = CudaStd::convert_string_arr_to_char_arr(vocab_list,num_unique_words,vocab_list_total_char_count,indexes_string_start_vocab);

	//Add the last index in the indexes lists
	d_cl_index_start[total_word_count]=total_char_count;
	d_vl_index_start[num_unique_words]=vocab_list_total_char_count;

	//copy Vocab list, vocab indexes, data indexes, char list
	hipMemcpy(d_cl,h_cl,total_char_count*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_vl,h_vl,vocab_list_total_char_count*sizeof(char),hipMemcpyHostToDevice);

	//copying indexes lists
	hipMemcpy(d_vl_index_start,indexes_string_start_vocab,num_unique_words*sizeof(int)+sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_table_indexes,temp_hashLists,num_unique_words*sizeof(int),hipMemcpyHostToDevice);

	hipMemcpy(d_cl_index_start,indexes_string_start_data,total_word_count*sizeof(int)+sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_positions,temp_positions,total_word_count*sizeof(int),hipMemcpyHostToDevice);

	construct_feature_vector_kernel<<<blocks,threads>>>(d_fv,d_cl,d_vl,d_cl_index_start,d_table_indexes,d_vl_index_start,d_positions);

	hipMemcpy(feature_vector,d_fv,total_word_count*sizeof(int),hipMemcpyDeviceToHost);
#ifndef CUDA_FEATURE_VECTOR
    // loop over every document
    for(int i=0;i<number_documents;i++)
    {
        for(int j=0;j<documents_size[i];j++)
        {
            // check if word can be used as feature word
            if(!check_if_feature(data_list[i][j]))
			{
				position = -1;
			}
			else
			{
				// calculate index
				hashIndex = SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
				// get the position of the word int he vocablist
				position = hash_list[hashIndex].getPositionValue(data_list[i][j]);
			}

			feature_vector[documents_indexes[i] + j] = position;
            
			//printf("%i ",feature_vector[get_document_index(documents_size,i) + j]);
        }
		//printf("\n");
		//printf("\n");
        
    }

	
	/***** DEALLOCATE UNUSED ARRAYS *****/
	//for(int i = 0;i<HASH_TABLE_SIZE;i++)
	//{
	//	hash_list[i].Destroy();
	//}

	//delete[] hash_list;
	#endif
    printf("Ended Feature Construction\n");
    
}
#endif