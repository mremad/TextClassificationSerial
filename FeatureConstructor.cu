#include "hip/hip_runtime.h"
#include "FeatureConstructor.h"
#include "Cfg.h"

using namespace std;

#undef get16bits
#if (defined(__GNUC__) && defined(__i386__)) || defined(__WATCOMC__) \
|| defined(_MSC_VER) || defined (__BORLANDC__) || defined (__TURBOC__)
#define get16bits(d) (*((const uint16_t *) (d)))
#endif

#if !defined (get16bits)
#define get16bits(d) ((((uint32_t)(((const uint8_t *)(d))[1])) << 8)\
+(uint32_t)(((const uint8_t *)(d))[0]) )
#endif

 

dataCollection dataCollection_new()
{
	 dataCollection a;

  return a;
}

FeatureConstructor::FeatureConstructor()
{
    
}

void FeatureConstructor::process_data_list(string** data_list,int* documents_size, int number_documents)
{
	total_char_count=0;
	total_word_count=0;
	for(int i=0;i<number_documents;i++)
	{
		total_word_count+=documents_size[i];
		for(int j=0;j<documents_size[i];j++)
		{
			total_char_count+=data_list[i][j].length();
		}
	}
}

int calculate_table_size(int* documents_size, int number_documents)
{
    int result=0;
    for(int i=0;i<number_documents;i++)
        result+=documents_size[i];
    return  result;
}

FeatureConstructor::FeatureConstructor(int* document_size, int number_documents)
{
	
    label_list= new string[MAX_NUM_LABELS];
    num_labels=0;
	num_unique_words=0;
	documents_size = document_size;

    label_list= new string[MAX_NUM_LABELS];
    hash_list= new LinkedList[HASH_TABLE_SIZE];
    max_List_Size=0;
	for(int i=0;i<HASH_TABLE_SIZE;i++)
    {
        hash_list[i]= *new LinkedList();
    }

}

FeatureConstructor::FeatureConstructor(int* document_size, int number_documents,int total_characters_count,int total_words_count)
{
    num_labels=0;
    num_unique_words=0;
	documents_size = document_size;

    total_char_count=total_characters_count;
	total_words_count= total_words_count;

    label_list= new string[MAX_NUM_LABELS];
    hash_list= new LinkedList[HASH_TABLE_SIZE];
    max_List_Size=0;
    for(int i=0;i<HASH_TABLE_SIZE;i++)
    {
        hash_list[i]= *new LinkedList();
    }
}

bool FeatureConstructor::check_if_feature(string word)
{
    bool check = true;
    
    if(word.length() <= 3)
    {
        check = false;
    }
    else if(word == "though" || word == "they" || word == "that" || word == "this" || word == "there" || word == "were"
            || word == "than" || word == "rather" || word == "from" || word == "most")
        check = false;
    
    return check;
}

int FeatureConstructor::hash_str(string word, int tableSize)
{
    int h = 31 /* also prime */;
    
    char * charList= (char *)malloc(word.length()*sizeof(char));
    for(int i=0;i<word.length();i++)
        charList[i]= word[i];
    
    
    while (*charList) {
       // h = (h * A) ^ (charList[0] * B);
        charList++;
    }
    return h%tableSize; // or return h % C;
}

void FeatureConstructor::extract_labels(string** data_list, int number_documents)
{
	for(int i=0;i<number_documents;i++)
    {
		bool found=false;
        // loop over all discovered labes see if it already added or not
        for(int l=0;l<num_labels; l++)
        {
            if (data_list[i][0] == label_list[l])
            {
                // if found, set the flag to true and replace the lablel with its index in the labellist
                found= true;
            }
        }
        
        // if the label isn't found in the label list
        if(!found && data_list[i][0] != "")
        {
            string s = data_list[i][0];
            printf("Index: %d\tLabel: %s\tDoc: %d\n",num_labels,data_list[i][0].c_str(),i);
            // add it to the list
            label_list[num_labels]= data_list[i][0];
            
            // increment the number of labels
            num_labels++;
            
        }
    }
    printf("Labels found: %d\n",num_labels);
}

void FeatureConstructor::extract_documents_indexes(int* documents_size, int number_documents)
{
	documents_indexes = (int*)malloc(sizeof(int)*number_documents);

	for(int i = 0;i < number_documents;i++)
	{
		documents_indexes[i] = get_document_index(documents_size,i);
	}
}

//Builds a list of all unique words in vocab_list from data_list
void FeatureConstructor::extract_vocab(string** data_list,int* documents_size, int number_documents)
{
    
    // loop over all documents and extract all labels
    printf("Began Vocab Extraction\n");
    
	extract_labels(data_list,number_documents);

    // loop over documents
    for(int i=0;i<number_documents; i++)
    {
        if( ((float)i/number_documents) == ((float)80/100) )
            printf("80%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)60/100))
            printf("60%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)40/100))
            printf("40%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)20/100))
            printf("20%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)10/100))
            printf("10%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(i == number_documents - 1)
            printf("100%% Completed: %d Unique words found: %d\n",i,num_unique_words);
            
        // loop over all words in this document
        for(int j=1;j<documents_size[i];j++)
        {
            if(!check_if_feature(data_list[i][j]))
                continue;
            
            // calculate the hash index
            int hashIndex= SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
            // if the word wasn't already added to the list in the specified index
            if(!hash_list[hashIndex].Exists(data_list[i][j]))
            {
                // append word to the list
                hash_list[hashIndex].Append((data_list[i][j]),num_unique_words);
				//printf("%s ",data_list[i][j]);
                num_unique_words++;
            }
           
        }
    }

	//printf("\n");
    
    printf("Ended Vocab Extraction\n");
}

int FeatureConstructor::get_index_for_label(string label)
{
    int index = -1;
    
    for(int i = 0;i < num_labels;i++)
    {
        if(label == label_list[i])
        {
            index = i;
            break;
        }
    }
    
    return index;
}

void FeatureConstructor::convert_labels_integers(string ** data_list, int number_documents)
{
    for(int i = 0;i<number_documents;i++)
    {
        documents_labels[i] = get_index_for_label(data_list[i][0]);
    }
}

int FeatureConstructor::get_document_index(int* documents_size, int doc_num)
{
	int index = 0;

	for(int i = 0;i<doc_num;i++)
	{
		index += documents_size[i];
	}

	return index;
}

//Builds feature vectors for all documents in document_feature_vectors from data_list
void FeatureConstructor::construct_feature_vectors(string** data_list,int* documents_size, int number_documents)
{
    printf("Began Feature Construction\n");
    int hashIndex, position;
	int totalSize = calculate_table_size(documents_size,number_documents);

    // set the number of rows to be equal number of documents
    feature_vector =  (int*)malloc(sizeof(int)*totalSize);
    documents_labels = (int*)malloc(sizeof(int)*number_documents);

    convert_labels_integers(data_list, number_documents);
	extract_documents_indexes(documents_size, number_documents);
    
    // loop on every row and set number of columns to be equal of number of unique words
    for(int i=0;i<totalSize;i++)
    {
		feature_vector[i]=0;
    }

    // loop over every document
    for(int i=0;i<number_documents;i++)
    {
        for(int j=0;j<documents_size[i];j++)
        {
            // check if word can be used as feature word
            if(!check_if_feature(data_list[i][j]))
			{
				position = -1;
			}
			else
			{
				// calculate index
				hashIndex = SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
				// get the position of the word int he vocablist
				position = hash_list[hashIndex].getPositionValue(data_list[i][j]);
			}

			feature_vector[documents_indexes[i] + j] = position;
            
			//printf("%i ",feature_vector[get_document_index(documents_size,i) + j]);
        }
		//printf("\n");
		//printf("\n");
        
    }

	
	/***** DEALLOCATE UNUSED ARRAYS *****/
	//for(int i = 0;i<HASH_TABLE_SIZE;i++)
	//{
	//	hash_list[i].Destroy();
	//}

	//delete[] hash_list;

    printf("Ended Feature Construction\n");
    
}

__device__ int hash_str(const char* s, int str_length) 
{ 
	unsigned h = 31 /* also prime */; 

	for(int i = 0; i < str_length;i++) 
	{ 
		h = (h * 54059 ) ^ (s[0] * 76963); 
	} 

	return h % HASH_TABLE_SIZE; 
}

__device__ uint32_t hash_inc(const char * data, int len, uint32_t hash)
{
    uint32_t tmp;
    int rem;
    
    if (len <= 0 || data == NULL) return 0;
    
    rem = len & 3;
    len >>= 2;
    
    /* Main loop */
    for (;len > 0; len--) {
        hash  += get16bits (data);
        tmp    = (get16bits (data+2) << 11) ^ hash;
        hash   = (hash << 16) ^ tmp;
        data  += 2*sizeof (uint16_t);
        hash  += hash >> 11;
    }
    
    /* Handle end cases */
    switch (rem) {
        case 3: hash += get16bits (data);
            hash ^= hash << 16;
            hash ^= data[sizeof (uint16_t)] << 18;
            hash += hash >> 11;
            break;
        case 2: hash += get16bits (data);
            hash ^= hash << 11;
            hash += hash >> 17;
            break;
        case 1: hash += *data;
            hash ^= hash << 10;
            hash += hash >> 1;
    }
    
    /* Force "avalanching" of final 127 bits */
    hash ^= hash << 3;
    hash += hash >> 5;
    hash ^= hash << 4;
    hash += hash >> 17;
    hash ^= hash << 25;
    hash += hash >> 6;
    
	return hash% HASH_TABLE_SIZE;
} 
__device__ int compare(char* a, char* b, int length_a)
{
  for(int i=0;i<length_a;i++)
  {
	  if(a[i]!=b[i])
		return 0;
  }
	  if(b[length_a]=='&')
		 return 1;
	  else
	  return 0;
}
//***********************TEMP TO BE REMOVED********************
struct  data_collection{
  char* char_data_list;
  int*  data_start_indexes;
  char* hash_array; 
  int*  words_per_hash_row;
  int* vocab_list_indices;
  int total_word_count;
  int total_char_count;
  int hash_table_size;
  int hash_row_size;
  int hash_word_size;

} ;
//******************************************************************
//*************************SHAABAN CODE******************************
__global__ void construct_feature_vector_kernel (int* d_fv, data_collection d_data, char* vocab_list)
{
	int threadId = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(threadId >= d_data.total_word_count)
		return;
	
	int my_word_start=d_data.data_start_indexes[threadId];
	//char* my_word=&d_data.char_data_list [my_word_start];
	int my_word_length;
	if(threadId+1<d_data.total_word_count)
		 my_word_length=d_data.data_start_indexes[threadId+1]-d_data.data_start_indexes[threadId];
	else
		 my_word_length=d_data.total_char_count-d_data.data_start_indexes[threadId];

	int index_list=hash_str(&d_data.char_data_list [my_word_start],my_word_length);
	int vocab_list_index=index_list*d_data.hash_row_size*d_data.hash_word_size;
	int vocab_list_length=d_data.words_per_hash_row[index_list]*d_data.hash_word_size;

	for(int i=vocab_list_index;(i<vocab_list_length+vocab_list_index) && (my_word_start+my_word_length < d_data.total_char_count );i+=d_data.hash_word_size)
	{

		if(compare(&d_data.char_data_list [my_word_start],&d_data.hash_array[i],my_word_length)==1)
		{
			d_fv[threadId]=i;
			break;
		}
	}
	
}

#ifdef CUDA_FEATURE_VECTOR
void FeatureConstructor::construct_feature_vectors(string** data_list,int* documents_size, int number_documents,int total_char_count,int total_word_count)
{
    printf("Began Feature Construction\n");
    int hashIndex, position;
	//int totalSize = calculate_table_size(documents_size,number_documents);

    // set the number of rows to be equal number of documents
    feature_vector =  (int*)malloc(sizeof(int)*total_word_count);
    documents_labels = (int*)malloc(sizeof(int)*number_documents);

    convert_labels_integers(data_list, number_documents);
	extract_documents_indexes(documents_size, number_documents);
    
    // loop on every row and set number of columns to be equal of number of unique words
    for(int i=0;i<total_word_count;i++)
    {
		feature_vector[i]=-1;
    }
	const int THREADS_Y = 1;
	const int THREADS_X = 512;
	const int BLOCKS_X = ceil(total_word_count/(float)THREADS_X);
	const int BLOCKS_Y = 1;

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int*  d_fv; //holding results
	char* d_cl;
	int* d_cl_index_start;
	char* d_vl;
	int* words_per_row;


	//******TEMP FOR VOCAB COMPUTATION
	/*int vocab_list_total_char_count=0;
	for(int i=0;i<num_unique_words;i++)
	{
		vocab_list_total_char_count+= vocab_list[i].length();
	}
	char* vocab_temp_list= (char*)malloc(sizeof(char)*num_unique_words*100);
	int* vocab_words_per_row_temp=(int*)malloc(sizeof(int)*2000);
	for(int i=0;i<num_unique_words;i++)
	{
		for(int j=0;j<vocab_list[i].length();j++)
			vocab_temp_list[(i*100)+j]=vocab_list[i].at(j);
		vocab_temp_list[(i*100)+vocab_list[i].length()]='&';
	}
	for(int i=0;i<2000;i++)
	{
		vocab_words_per_row_temp[i]=60;
	}*/
	//**************


	//allocate feature vector, char list(data list->chars), vocab list->char,index for vocab list and data list
	hipMalloc((void**)&d_fv,total_word_count*sizeof(int));

	hipMalloc((void**)&d_cl,total_char_count*sizeof(char));
	hipMalloc((void**)&d_vl,HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE*sizeof(char));

	hipMalloc((void**)&words_per_row,HASH_TABLE_SIZE*sizeof(int));

	hipMalloc((void**)&d_cl_index_start,total_word_count*sizeof(int));



	//convert the datalist to char list and extract both indexes
	string* h_dl;//temp to hold the data into 1D string array
	//convert 2D array to 1D array of strings
	h_dl = CudaStd::convert_2d_to_1d_string(data_list, number_documents,documents_size,total_word_count);
	int* indexes_string_start_data=(int*)malloc(sizeof(int)*total_word_count);//indexes for the start and length of chars
	//int* indexes_string_length_data=(int*)malloc(sizeof(int)*total_word_count);
	//convert 1D array of strings to 1D array of chars
	char* h_cl = CudaStd::convert_string_arr_to_char_arr(h_dl,total_word_count,total_char_count,indexes_string_start_data);
//	indexes_string_start_data[0]=0;
	//for(int i=1;i<total_word_count;i++)
//	{
	//	indexes_string_start_data[i]=h_dl[i].length()+indexes_string_start_data[i-1];
	//}
	//for(int i=0;i<500;i++)
	//{
	//	printf("%c",vocab_temp_list[i]);
	//}
	//for(int i=0;i<100;i++)
	//{
	//	printf("%d ",indexes_string_start_data[i]);
//	}

	
	//processing the vocab list
	//int* indexes_string_start_vocab=(int*)malloc(sizeof(int)*num_unique_words);//indexes for the start and length of chars
	//int* indexes_string_length_vocab=(int*)malloc(sizeof(int)*num_unique_words);
	//convert 1D array of strings to 1D array of chars
	//char* h_vl = CudaStd::convert_string_arr_to_char_arr(vocab_list,num_unique_words,vocab_list_total_char_count,indexes_string_start_vocab);
	

	


	//constants for the data structure
	data_collection data;
	data.char_data_list=d_cl;
	data.data_start_indexes=d_cl_index_start;
	data.hash_array=d_vl;
	data.hash_row_size=HASH_ROW_SIZE;
	data.hash_table_size=HASH_TABLE_SIZE;
	data.hash_word_size=HASH_WORD_SIZE;
	data.total_char_count=total_char_count;
	data.total_word_count=total_word_count;
	data.words_per_hash_row=words_per_row;
	


	//Add the last index in the indexes lists
	//d_cl_index_start[total_word_count]=total_char_count;
	//d_vl_index_start[num_unique_words]=vocab_list_total_char_count;

	//copy Vocab list, vocab indexes, data indexes, char list
	hipMemcpy(d_cl,h_cl,total_char_count*sizeof(char),hipMemcpyHostToDevice);

	hipMemcpy(d_vl,h_hash_array,HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE*sizeof(char),hipMemcpyHostToDevice);

	//copying indexes lists
	hipMemcpy(words_per_row,h_words_per_hash_row,HASH_TABLE_SIZE*sizeof(int),hipMemcpyHostToDevice);

	hipMemcpy(d_cl_index_start,indexes_string_start_data,total_word_count*sizeof(int),hipMemcpyHostToDevice);
	char* myVocab;
	char* host=(char*)malloc(sizeof(char)*500);
	for(int i=0;i<500;i++)
	{
		host[i]='x';
	}
	hipMalloc((void**)&myVocab,sizeof(char)*500);
 	construct_feature_vector_kernel<<<blocks,threads>>>(d_fv,data,myVocab);
	if ( hipSuccess != hipGetLastError() )
    printf( "Error!\n" );

	hipMemcpy(feature_vector,d_fv,total_word_count*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(host,myVocab,sizeof(char)*500,hipMemcpyDeviceToHost);
	hipFree(d_cl);
	hipFree(d_vl);
	hipFree(words_per_row);
#ifndef CUDA_FEATURE_VECTOR
    // loop over every document
    for(int i=0;i<number_documents;i++)
    {
        for(int j=0;j<documents_size[i];j++)
        {
            // check if word can be used as feature word
            if(!check_if_feature(data_list[i][j]))
			{
				position = -1;
			}
			else
			{
				// calculate index
				hashIndex = SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
				// get the position of the word int he vocablist
				position = hash_list[hashIndex].getPositionValue(data_list[i][j]);
			}

			feature_vector[documents_indexes[i] + j] = position;
            
			//printf("%i ",feature_vector[get_document_index(documents_size,i) + j]);
        }
		//printf("\n");
		//printf("\n");
        
    }

	
	/***** DEALLOCATE UNUSED ARRAYS *****/
	//for(int i = 0;i<HASH_TABLE_SIZE;i++)
	//{
	//	hash_list[i].Destroy();
	//}

	//delete[] hash_list;
	#endif
	for(int i=0;i<600;i++)
	{
		printf("%d ",feature_vector[i]);
	}
	/*for(int i=0;i<100;i++)
	{
		printf("%c",host[i]);
	}*/
    printf("Ended Feature Construction\n");
    
}
#endif
//*******************************************************************



__device__ int atomic_add(int *address, int value)
{
	int result=atomicAdd(address,value);
	return result;
}

__global__ void extract_vocab_kernel(dataCollection d_data  )
{
	int threadId = threadIdx.x + blockIdx.x*blockDim.x;

	if(threadId >= d_data.total_word_count)
		return;

	char *word;
	int startIndex, wordLength, hashValue;
	bool wordFound=false;

	// get the start index of the current word
	startIndex=d_data.d_indexes_string_start[threadId];
	word= &d_data.d_charDataList[startIndex];

	//check if the current word is not the last word
	if((threadId+1)< d_data.total_word_count)
	{
		// get the word length = start of next word - start of current word
		wordLength= d_data.d_indexes_string_start[threadId+1]- d_data.d_indexes_string_start[threadId];
	}
	else
	{
		// case the word is the last word:get word length = total number of chars - start of current word
		wordLength= d_data.total_char_count- d_data.d_indexes_string_start[threadId];
	}


	// get the hash value of the current word
	hashValue= hash_str(word,wordLength);
	
	// Loop in the words in the hash table row with same hash value 
	// to see if word exists or not
	for(int j=0;j< d_data.d_words_per_hash_row[hashValue];j++)
	{	
		// calculate the start of the word in hash table
		int wordStartinHashTable=hashValue*d_data.hash_row_size*d_data.hash_word_size+ j*d_data.hash_word_size; 
		if(compare(&d_data.d_charDataList[startIndex],&d_data.d_HashArray[wordStartinHashTable], wordLength))
		{
			wordFound= true;
			break;
		}
	}

	if(!wordFound)
	{
		// update the number of words in this hash row
		int oldNumOfWords=  atomic_add(&d_data.d_words_per_hash_row[hashValue],1);
		// caluclat the  new index where the word will be written into 
		int hashArrayIndex=hashValue*d_data.hash_row_size*d_data.hash_word_size+ oldNumOfWords*d_data.hash_word_size;
		
		// copy data from data list to hash value
		for(int k=0;k<wordLength;k++)
		{
			d_data.d_HashArray[hashArrayIndex+k]= d_data.d_charDataList[startIndex+k];
		}
		d_data.d_HashArray[hashArrayIndex+wordLength]='&';
	}


	

}


void FeatureConstructor::extract_vocab(string** data_list,int* documents_size, int number_documents, int total_word_count, int total_char_count)
{
     //loop over all documents and extract all labels
    printf("Began Labels Extraction\n");
	extract_labels(data_list,number_documents);
    printf("End of  Labels Extraction\n");

    printf("Began Vocab Extraction\n");

	// initialize variables
	int* indexes_string_start_data;
	char* h_charDataList;
	string* h_dl;
	char *d_hash_array, *d_charDataList;
	int *d_words_per_hash_row, *d_indexes_string_start_data  ;

    //convert 2D array to 1D array of strings
	 h_dl = CudaStd::convert_2d_to_1d_string(data_list, number_documents,documents_size,total_word_count);
	// get start index of each word
     indexes_string_start_data=(int*)malloc(sizeof(int)*total_word_count);//indexes for the start and length of chars
    //convert 1D array of strings to 1D array of chars
	 h_charDataList = CudaStd::convert_string_arr_to_char_arr(h_dl,total_word_count,total_char_count,indexes_string_start_data);

	 for(int i = 0;i<20;i++)
		 printf("%c",h_charDataList[i]);
	 printf("\n");

	 //Allocate memory to arrays on the host
	 // size of h_hash_array= (number of rows in hash table)* (each row size)* (size of each word) 
	h_hash_array=(char*) malloc(sizeof(char)*HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE);
	h_words_per_hash_row= (int*)malloc(sizeof(int)*HASH_TABLE_SIZE);

	// initialize h_words_per_hash_row with zeros
	for(int i=0;i<HASH_TABLE_SIZE;i++)
	{
		h_words_per_hash_row[i]=0;
	}
	
	//allocate memory on the device
	hipMalloc((void**)&d_charDataList,sizeof(char)*total_char_count);
	hipMalloc((void**)&d_indexes_string_start_data,sizeof(int)*total_word_count);
	hipMalloc((void**)&d_hash_array,sizeof(char)*HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE);
	hipMalloc((void**)&d_words_per_hash_row,sizeof(int)*HASH_TABLE_SIZE);

	// copy data from device to host
	hipMemcpy(d_charDataList,h_charDataList,total_char_count*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_indexes_string_start_data,indexes_string_start_data,total_word_count*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_words_per_hash_row,h_words_per_hash_row,HASH_TABLE_SIZE*sizeof(int),hipMemcpyHostToDevice);
    
	
    const int THREADS_X = 512;
    const int THREADS_Y = 1;
    const int BLOCKS_X = ceil(total_word_count/(float)THREADS_X);
    const int BLOCKS_Y = 1;

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	// use the dataCollectin struct to pass arguments to the kernel
	dataCollection d_data;
	d_data.d_charDataList= d_charDataList;
	d_data.d_indexes_string_start= d_indexes_string_start_data;
	d_data.d_HashArray= d_hash_array;
	d_data.d_words_per_hash_row= d_words_per_hash_row;
	d_data.total_char_count= total_char_count;
	d_data.total_word_count= total_word_count;
	d_data.hash_table_size= HASH_TABLE_SIZE;
	d_data.hash_row_size= HASH_ROW_SIZE;
	d_data.hash_word_size= HASH_WORD_SIZE;

	// run the kernel 
	extract_vocab_kernel<<<blocks,threads>>>(d_data);

	// copy the data back from device to host
	hipMemcpy(h_hash_array,d_hash_array,HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE*sizeof(char),hipMemcpyDeviceToHost);
	hipMemcpy(h_words_per_hash_row,d_words_per_hash_row,sizeof(int)*HASH_TABLE_SIZE,hipMemcpyDeviceToHost);
 
	// calculate numver of unique words 
	for(int i=0;i<HASH_TABLE_SIZE;i++)
	{
		num_unique_words+= h_words_per_hash_row[i];
	}

	// Free memory on the device
	hipFree(d_data.d_charDataList);
	hipFree(d_data.d_HashArray);
	hipFree(d_data.d_indexes_string_start);
	hipFree(d_data.d_words_per_hash_row);

	// free memory on the host
	free(h_charDataList);
	free(indexes_string_start_data);
	//delete(h_dl);

    printf("End of Vocab Extraction\n");

}