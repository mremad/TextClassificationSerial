#include "hip/hip_runtime.h"
#include "FeatureConstructor.h"


using namespace std;

dataCollection dataCollection_new()
{
	 dataCollection a;

  return a;
}

FeatureConstructor::FeatureConstructor()
{
    
}

void FeatureConstructor::process_data_list(string** data_list,int* documents_size, int number_documents)
{
	total_char_count=0;
	total_word_count=0;
	for(int i=0;i<number_documents;i++)
	{
		total_word_count+=documents_size[i];
		for(int j=0;j<documents_size[i];j++)
		{
			total_char_count+=data_list[i][j].length();
		}
	}

	
}

int calculate_table_size(int* documents_size, int number_documents)
{
    int result=0;
    for(int i=0;i<number_documents;i++)
        result+=documents_size[i];
    return  result;
}

FeatureConstructor::FeatureConstructor(int* document_size, int number_documents)
{
	
    label_list= new string[MAX_NUM_LABELS];
    num_labels=0;
	num_unique_words=0;
	documents_size = document_size;
    label_list= new string[MAX_NUM_LABELS];
    hash_list= new LinkedList[HASH_TABLE_SIZE];
    max_List_Size=0;
	for(int i=0;i<HASH_TABLE_SIZE;i++)
    {
        hash_list[i]= *new LinkedList();
    }

}

int FeatureConstructor::hash_str(string word, int tableSize)
{
    int h = 31 /* also prime */;
    
    char * charList= (char *)malloc(word.length()*sizeof(char));
    for(int i=0;i<word.length();i++)
        charList[i]= word[i];
    
    
    while (*charList) {
       // h = (h * A) ^ (charList[0] * B);
        charList++;
    }
    return h%tableSize; // or return h % C;
}

void FeatureConstructor::extract_labels(string** data_list, int number_documents)
{
	for(int i=0;i<number_documents;i++)
    {
		bool found=false;
        // loop over all discovered labes see if it already added or not
        for(int l=0;l<num_labels; l++)
        {
            if (data_list[i][0] == label_list[l])
            {
                // if found, set the flag to true and replace the lablel with its index in the labellist
                found= true;
            }
        }
        
        // if the label isn't found in the label list
        if(!found && data_list[i][0] != "")
        {
            string s = data_list[i][0];
            printf("Index: %d\tLabel: %s\tDoc: %d\n",num_labels,data_list[i][0].c_str(),i);
            // add it to the list
            label_list[num_labels]= data_list[i][0];
            
            // increment the number of labels
            num_labels++;
            
        }
    }
    printf("Labels found: %d\n",num_labels);
}

void FeatureConstructor::extract_documents_indexes(int* documents_size, int number_documents)
{
	documents_indexes = (int*)malloc(sizeof(int)*number_documents);

	for(int i = 0;i < number_documents;i++)
	{
		documents_indexes[i] = get_document_index(documents_size,i);
	}
}

//Builds a list of all unique words in vocab_list from data_list
void FeatureConstructor::extract_vocab(string** data_list,int* documents_size, int number_documents)
{
    
    // loop over all documents and extract all labels
    printf("Began Vocab Extraction\n");
    
	extract_labels(data_list,number_documents);

    // loop over documents
    for(int i=0;i<number_documents; i++)
    {
        if( ((float)i/number_documents) == ((float)80/100) )
            printf("80%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)60/100))
            printf("60%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)40/100))
            printf("40%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)20/100))
            printf("20%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(((float)i/number_documents) == ((float)10/100))
            printf("10%% Completed: %d Unique words found: %d\n",i,num_unique_words);
        else if(i == number_documents - 1)
            printf("100%% Completed: %d Unique words found: %d\n",i,num_unique_words);
            
        // loop over all words in this document
        for(int j=0;j<documents_size[i];j++)
        {
            // calculate the hash index
            int hashIndex= SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
            // if the word wasn't already added to the list in the specified index
            if(!hash_list[hashIndex].Exists(data_list[i][j]))
            {
                // append word to the list
                hash_list[hashIndex].Append((data_list[i][j]),num_unique_words);

				//printf("%s ",data_list[i][j]);
                num_unique_words++;
            }
           
        }
    }

	//printf("\n");
    
    printf("Ended Vocab Extraction\n");
}

int FeatureConstructor::get_index_for_label(string label)
{
    int index = -1;
    
    for(int i = 0;i < num_labels;i++)
    {
        if(label == label_list[i])
        {
            index = i;
            break;
        }
    }
    
    return index;
}

void FeatureConstructor::convert_labels_integers(string ** data_list, int number_documents)
{
    for(int i = 0;i<number_documents;i++)
    {
        documents_labels[i] = get_index_for_label(data_list[i][0]);
    }
}

int FeatureConstructor::get_document_index(int* documents_size, int doc_num)
{
	int index = 0;

	for(int i = 0;i<doc_num;i++)
	{
		index += documents_size[i];
	}

	return index;
}

__device__ int hash_str(const char* s, int str_length) 
{ 
unsigned h = 31 /* also prime */; 

for(int i = 0; i < str_length;i++) 
{ 
h = (h * 54059 ) ^ (s[0] * 76963); 
} 

return h % HASH_TABLE_SIZE; 
}

__device__ int compare(char* a, char* b, int length_a)
{
  for(int i=0;i<length_a;i++)
  {
	  if(a[i]!=b[i])
		return 0;
  }
	  if(b[length_a]=='&')
		 return 1;
	  else
	  return 0;
}

// kernel for feature construction
__global__ void construct_feature_vector_kernel (int* d_fv, dataCollection d_data)
{
	//initialize threadId and a word from the data list for each thread
	int threadId = threadIdx.x + blockIdx.x*blockDim.x;
	if(threadId<d_data.total_word_count)
	{	
		d_fv[threadId]=-1;
		//each word has its starting index in the 1D char array saved in the data_start_indexes
		int my_word_start=d_data.d_indexes_string_start[threadId];

		//char* my_word=&d_data.char_data_list [my_word_start];
	
		int my_word_length;
		if(threadId+1<d_data.total_word_count)
			my_word_length=d_data.d_indexes_string_start[threadId+1]-d_data.d_indexes_string_start[threadId];
		else
			my_word_length=d_data.total_char_count-d_data.d_indexes_string_start[threadId];
		//calculate the hash function for this word and get the range in the vocab array to search in
		int index_list=hash_str(&d_data.d_charDataList [my_word_start],my_word_length);
		int vocab_list_index=index_list*d_data.hash_row_size*d_data.hash_word_size;
		int vocab_list_length=d_data.d_words_per_hash_row[index_list]*d_data.hash_word_size;

		//loop over the range from the vocab list and if the word matches a word stored in the vocab list save its index
		for(int i=vocab_list_index;i<(vocab_list_length+vocab_list_index) ;i+=d_data.hash_word_size)
		{
			if(compare(&d_data.d_charDataList [my_word_start],&d_data.d_HashArray[i],my_word_length)==1)
			{
				d_fv[threadId]=d_data.d_word_index_in_vocab[i];
				break;
			}
		}
	}
	
}

void FeatureConstructor::construct_feature_vectors(string** data_list,int* documents_size, int number_documents)
{
    
    printf("Began Feature Construction\n");
#ifndef CUDA_FEATURE_VECTOR
    int hashIndex, position;
	//int totalSize = calculate_table_size(documents_size,number_documents);

#endif
    feature_vector =  (int*)malloc(sizeof(int)*total_word_count);
    documents_labels = (int*)malloc(sizeof(int)*number_documents);

    convert_labels_integers(data_list, number_documents);
	extract_documents_indexes(documents_size, number_documents);
    

#ifndef CUDA_FEATURE_VECTOR
	// loop on every row and set number of columns to be equal of number of unique words
    for(int i=0;i<total_word_count;i++)
    {
		feature_vector[i]=0;
    }

    // loop over every document
    for(int i=0;i<number_documents;i++)
    {
        for(int j=0;j<documents_size[i];j++)
        {
				// calculate index
				hashIndex = SuperHash::create_hash(data_list[i][j], (int)data_list[i][j].length(),HASH_TABLE_SIZE);
				// get the position of the word int he vocablist
				position = hash_list[hashIndex].getPositionValue(data_list[i][j]);

			feature_vector[documents_indexes[i] + j] = position;
            
			//printf("%i ",feature_vector[get_document_index(documents_size,i) + j]);
        }
		//printf("\n");
		//printf("\n");
        
    }

	
	/***** DEALLOCATE UNUSED ARRAYS *****/
	//for(int i = 0;i<HASH_TABLE_SIZE;i++)
	//{
	//	hash_list[i].Destroy();
	//}

	//delete[] hash_list;

#endif

#ifdef CUDA_FEATURE_VECTOR
   
	const int THREADS_Y = 1;
	const int THREADS_X = 512;
	const int BLOCKS_X = ceil(total_word_count/(float)THREADS_X);
	const int BLOCKS_Y = 1;

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int*  d_fv; //holding results
	char* d_cl;
	int* d_cl_index_start;
	char* d_vl;
	int* words_per_row;
	int* d_word_index_in_vocab;


	//allocate feature vector, char list(data list->chars), vocab list->char,index for vocab list and data list
	hipMalloc((void**)&d_fv,total_word_count*sizeof(int));

	hipMalloc((void**)&d_word_index_in_vocab,HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE*sizeof(int));

	hipMalloc((void**)&d_cl,total_char_count*sizeof(char));
	hipMalloc((void**)&d_vl,HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE*sizeof(char));

	hipMalloc((void**)&words_per_row,HASH_TABLE_SIZE*sizeof(int));

	hipMalloc((void**)&d_cl_index_start,total_word_count*sizeof(int));

	

	//convert the datalist to char list and extract both indexes
	string* h_dl;//temp to hold the data into 1D string array
	//convert 2D array to 1D array of strings
	h_dl = CudaStd::convert_2d_to_1d_string(data_list, number_documents,documents_size,total_word_count);
	int* indexes_string_start_data=(int*)malloc(sizeof(int)*total_word_count);//indexes for the start and length of data strings
	//convert 1D array of strings to 1D array of chars
	char* h_cl = CudaStd::convert_string_arr_to_char_arr(h_dl,total_word_count,total_char_count,indexes_string_start_data);

	//constants for the data structure
	dataCollection data;
	data.d_charDataList=d_cl;
	data.d_indexes_string_start=d_cl_index_start;
	data.d_HashArray=d_vl;
	data.hash_row_size=HASH_ROW_SIZE;
	data.hash_table_size=HASH_TABLE_SIZE;
	data.hash_word_size=HASH_WORD_SIZE;
	data.total_char_count=total_char_count;
	data.total_word_count=total_word_count;
	data.d_words_per_hash_row=words_per_row;
	data.d_word_index_in_vocab=d_word_index_in_vocab;

	//copy Vocab list, vocab indexes, data indexes, char list
	hipMemcpy(d_cl,h_cl,total_char_count*sizeof(char),hipMemcpyHostToDevice);

	hipMemcpy(d_vl,h_hash_array,HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE*sizeof(char),hipMemcpyHostToDevice);

	hipMemcpy(d_word_index_in_vocab,h_word_index_in_vocab,HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE*sizeof(int),hipMemcpyHostToDevice);

	//copying indexes lists
	hipMemcpy(words_per_row,h_words_per_hash_row,HASH_TABLE_SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_cl_index_start,indexes_string_start_data,total_word_count*sizeof(int),hipMemcpyHostToDevice);
	
 	construct_feature_vector_kernel<<<blocks,threads>>>(d_fv,data);
	if ( hipSuccess != hipGetLastError() )
		printf("\n %s \n", hipGetErrorString(hipGetLastError()));
    //printf( "Error! Feature constructor parallel\n" );

	hipMemcpy(feature_vector,d_fv,total_word_count*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_cl);
	hipFree(d_vl);
	hipFree(words_per_row);
	hipFree(d_fv);
	hipFree(d_cl_index_start);
	hipFree(d_word_index_in_vocab);
	free(h_cl);
	free(indexes_string_start_data);

    printf("Ended Feature Construction\n");
   
#endif 
}

__device__ int atomic_add(int *address, int value)
{
	int result=atomicAdd(address,value);
	return result;
}

__device__ void shiftData( char* source, int data_length, int shift)
{

	// if shift is to the right, copy from the end first
	if(shift >0)
	{
		for(int i=data_length-1;i>0;i--)
		{
			source[i+shift]= source[i];
		}
	}
	else 
	{
		// if shift is to the left, copy from the start first
		for(int i=0;i<data_length;i++)
		{
			source[i+shift]= source[i];
		}

	}
	
}

__global__ void remove_duplicates_in_hashTable(dataCollection d_data)
{
	int threadId = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(threadId <d_data.hash_table_size)
	{
		for(int i=0;i<d_data.d_words_per_hash_row[threadId];i++)
		{
			int wordLength=0;
			int hashArrayIndex= threadId*d_data.hash_row_size*d_data.hash_word_size+ + i*d_data.hash_word_size;
			for(int j=0;j<d_data.hash_word_size;j++)
			{
				if(d_data.d_HashArray[hashArrayIndex+j]=='&')
					break;
				else
					wordLength++;
			}

			for(int k=i+1;k< d_data.d_words_per_hash_row[threadId];k++)
			{
				int comparedWordStart= threadId*d_data.hash_row_size*d_data.hash_word_size+ + k*d_data.hash_word_size;
				if(compare(&d_data.d_HashArray[hashArrayIndex] ,&d_data.d_HashArray[comparedWordStart], wordLength))
				{
					// check if compared word is the last word
					if(k == (d_data.d_words_per_hash_row[threadId]-1))
					{
						// just decrement the number of words in hash & ignore last word
						d_data.d_words_per_hash_row[threadId]--;
					}
					else
					{	
						// shift the next cells to the left & decrement the number of words per hash
						int startOfShiftedData= comparedWordStart+d_data.hash_word_size;
						int shiftedDataLength= (d_data.d_words_per_hash_row[threadId]-1-k)*d_data.hash_word_size;
						shiftData(&d_data.d_HashArray[startOfShiftedData],shiftedDataLength,-d_data.hash_word_size);
						d_data.d_words_per_hash_row[threadId]--;
						k--;
					}
				}
			}
		}
	}
	
}

__global__ void extract_vocab_kernel(dataCollection d_data  )
{
	int threadId = threadIdx.x + blockIdx.x*blockDim.x;
	if(threadId<d_data.total_word_count)
	{
		char *word;
		int startIndex, wordLength, hashValue;
		bool wordFound=false;

		// get the start index of the current word
		startIndex=d_data.d_indexes_string_start[threadId];
		word= &d_data.d_charDataList[startIndex];

		//check if the current word is not the last word
		if((threadId+1)< d_data.total_word_count)
		{
			// get the word length = start of next word - start of current word
			wordLength= d_data.d_indexes_string_start[threadId+1]- d_data.d_indexes_string_start[threadId];
		}
		else
		{
			// case the word is the last word:get word length = total number of chars - start of current word
			wordLength= d_data.total_char_count- d_data.d_indexes_string_start[threadId];
		}


		// get the hash value of the current word
		hashValue= hash_str(word,wordLength);
	
		// Loop in the words in the hash table row with same hash value 
		// to see if word exists or not
		for(int j=0;j< d_data.d_words_per_hash_row[hashValue];j++)
		{	
			// calculate the start of the word in hash table
			int wordStartinHashTable=hashValue*d_data.hash_row_size*d_data.hash_word_size+ j*d_data.hash_word_size; 
			if(compare(&d_data.d_charDataList[startIndex],&d_data.d_HashArray[wordStartinHashTable], wordLength))
			{
				wordFound= true;
				break;
			}
		}

		if(!wordFound)
		{
			// update the number of words in this hash row
			int oldNumOfWords=  atomic_add(&d_data.d_words_per_hash_row[hashValue],1);
			// caluclat the  new index where the word will be written into 
			int hashArrayIndex=hashValue*d_data.hash_row_size*d_data.hash_word_size+ oldNumOfWords*d_data.hash_word_size;
		
			// copy data from data list to hash value
			for(int k=0;k<wordLength;k++)
			{
				d_data.d_HashArray[hashArrayIndex+k]= d_data.d_charDataList[startIndex+k];
			}
			d_data.d_HashArray[hashArrayIndex+wordLength]='&';
		}
	}

	
	
}

void FeatureConstructor::extract_vocab(string** data_list,int* documents_size, int number_documents, int total_word_count, int total_char_count)
{
     //loop over all documents and extract all labels
    printf("Began Labels Extraction\n");
	extract_labels(data_list,number_documents);
    printf("End of  Labels Extraction\n");

    printf("Began Vocab Extraction\n");

	// initialize variables
	int* indexes_string_start_data;
	char* h_charDataList;
	string* h_dl;
	char *d_hash_array, *d_charDataList;
	int *d_words_per_hash_row, *d_indexes_string_start_data  ;

    //convert 2D array to 1D array of strings
	 h_dl = CudaStd::convert_2d_to_1d_string(data_list, number_documents,documents_size,total_word_count);
	// get start index of each word
     indexes_string_start_data=(int*)malloc(sizeof(int)*total_word_count);//indexes for the start and length of chars
    //convert 1D array of strings to 1D array of chars
	 h_charDataList = CudaStd::convert_string_arr_to_char_arr(h_dl,total_word_count,total_char_count,indexes_string_start_data);


	 //Allocate memory to arrays on the host
	 // size of h_hash_array= (number of rows in hash table)* (each row size)* (size of each word) 
	h_hash_array=(char*) malloc(sizeof(char)*HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE);
	h_words_per_hash_row= (int*)malloc(sizeof(int)*HASH_TABLE_SIZE);

	// initialize h_words_per_hash_row with zeros
	for(int i=0;i<HASH_TABLE_SIZE;i++)
	{
		h_words_per_hash_row[i]=0;
	}
	
	//allocate memory on the device
	hipMalloc((void**)&d_charDataList,sizeof(char)*total_char_count);
	hipMalloc((void**)&d_indexes_string_start_data,sizeof(int)*total_word_count);
	hipMalloc((void**)&d_hash_array,sizeof(char)*HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE);
	hipMalloc((void**)&d_words_per_hash_row,sizeof(int)*HASH_TABLE_SIZE);

	// copy data from device to host
	hipMemcpy(d_charDataList,h_charDataList,total_char_count*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_indexes_string_start_data,indexes_string_start_data,total_word_count*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_words_per_hash_row,h_words_per_hash_row,HASH_TABLE_SIZE*sizeof(int),hipMemcpyHostToDevice);
    
	
    const int THREADS_X = 512;
    const int THREADS_Y = 1;
    const int BLOCKS_X = ceil(total_word_count/(float)THREADS_X);
    const int BLOCKS_Y = 1;

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	// use the dataCollectin struct to pass arguments to the kernel
	dataCollection d_data;
	d_data.d_charDataList= d_charDataList;
	d_data.d_indexes_string_start= d_indexes_string_start_data;
	d_data.d_HashArray= d_hash_array;
	d_data.d_words_per_hash_row= d_words_per_hash_row;
	d_data.total_char_count= total_char_count;
	d_data.total_word_count= total_word_count;
	d_data.hash_table_size= HASH_TABLE_SIZE;
	d_data.hash_row_size= HASH_ROW_SIZE;
	d_data.hash_word_size= HASH_WORD_SIZE;

	// run the kernel 
	extract_vocab_kernel<<<blocks,threads>>>(d_data);
	const int numberOfBlocks= ceil(HASH_TABLE_SIZE/(float)THREADS_X);

	dim3 hash_blocks(numberOfBlocks,1);
	remove_duplicates_in_hashTable<<<hash_blocks,threads>>>(d_data);


	// copy the data back from device to host
	hipMemcpy(h_hash_array,d_hash_array,HASH_TABLE_SIZE*HASH_ROW_SIZE*HASH_WORD_SIZE*sizeof(char),hipMemcpyDeviceToHost);
	hipMemcpy(h_words_per_hash_row,d_words_per_hash_row,sizeof(int)*HASH_TABLE_SIZE,hipMemcpyDeviceToHost);
 
	// calculate numver of unique words 
	for(int i=0;i<HASH_TABLE_SIZE;i++)
	{
		num_unique_words+= h_words_per_hash_row[i];
	}

	/*for(int i=0;i<HASH_TABLE_SIZE;i++)
	{
		for(int j=0;j<h_words_per_hash_row[i];j++)
		{
			for(int k=0;k<HASH_WORD_SIZE;k++)
			{
				int index= i*HASH_ROW_SIZE*HASH_WORD_SIZE+ j*HASH_WORD_SIZE;
				if(h_hash_array[index+k]== '&')
					break;

				printf("%c",h_hash_array[index+k]);
			}
			 printf(" - ");
		}

		printf("\n End of Row %d ------------------------------------------------------ \n" ,i);
	}*/
	
	//hold index in vocab list
	// Row size = 10 words
	// word size = 10 chars
	//3 words  1st row :(0)messi  ,(10) ronaldo  (20) treka
	//2 words   2nd row ; (0) emad , (10) shaaban
	//0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0
	//
 
	
	h_word_index_in_vocab=(int*)malloc(sizeof(int)*HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE);

		for(int j=0;j<HASH_ROW_SIZE*HASH_WORD_SIZE*HASH_TABLE_SIZE;j++)
		{
			h_word_index_in_vocab[j]=-1;
		}
	//calculate index of each vocab word in the vocab list
	int index_in_h_hash_array=0;
	for(int i=0;i<HASH_TABLE_SIZE;i++)
		for(int j=0;j<h_words_per_hash_row[i];j++)
		{
			h_word_index_in_vocab[j*HASH_WORD_SIZE+i*HASH_ROW_SIZE*HASH_WORD_SIZE]=index_in_h_hash_array;
			index_in_h_hash_array++;
		}
		
	// Free memory on the device
	hipFree(d_data.d_charDataList);
	hipFree(d_data.d_HashArray);
	hipFree(d_data.d_indexes_string_start);
	hipFree(d_data.d_words_per_hash_row);

	// free memory on the host
	free(h_charDataList);
	free(indexes_string_start_data);
	//delete(h_dl);

    printf("End of Vocab Extraction\n");

}