#include "hip/hip_runtime.h"
#include "NaiveBayesClassifier.h"
#include "ConsolePrint.h"

#define N_INF -1000000000

NaiveBayesClassifier::NaiveBayesClassifier(int number_labels,int number_unique_words)
{
	first_parameter = (double**)malloc(sizeof(double*)*number_labels);
	second_parameter = (double*)malloc(sizeof(double)*number_labels);

	for(int i = 0;i<number_labels;i++)
		first_parameter[i] = (double*)malloc(sizeof(double)*number_unique_words);


}

#ifdef CUDA_LIKELIHOOD

__device__ double calculate_single_word_label_occurence(int * feature_vectors,int * documents_size, int * documents_indexes, int * docs_labels, int word_index, int number_documents, int label,int num_labels, double* weights)
{
	double result = 0;

	for(int i = 0;i<number_documents;i++)
	{
		if(label == docs_labels[i])
		{
			for(int j = 0;j < documents_size[i];j++)
			{
				if(feature_vectors[documents_indexes[i] + j] == word_index)
					result++;
			}

		}
		else if(docs_labels[i] == -1)
		{
			for(int j = 0;j < documents_size[i];j++)
			{
				if(feature_vectors[documents_indexes[i] + j] == word_index)
				{
						result += (weights[i*num_labels + label]*1);
				}
			}
		}
			
	}


	return result;

}

__global__ void calculate_occurences_kernel(int * d_fv, double* d_fp ,int * documents_size, int * documents_indexes, int * docs_labels, size_t pitch_fp, int num_docs, int num_unique_words, int num_labels, double* weights)
{
	
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y;
	
	if(x < num_unique_words && y < num_labels)
	{
		((double*)((char*)d_fp+y*pitch_fp))[x] = calculate_single_word_label_occurence(d_fv,documents_size,documents_indexes,docs_labels,x,num_docs,y,num_labels,weights);
	}
}

__global__ void calculate_likelihood_kernel(double* d_fp,double* d_occ, size_t pitch_occ, size_t pitch_fp, int num_unique_words, int num_labels)
{
	
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y;
	
	if(x < num_unique_words && y < num_labels)
	{
		double all_label_occ = 0;

		for(int i = 0;i < num_unique_words;i++)
			all_label_occ += ((double*)((char*)d_occ+y*pitch_occ))[i];

		((double*)((char*)d_fp+y*pitch_fp))[x] = (((double*)((char*)d_occ+y*pitch_occ))[x] + 1)/
					(double)(all_label_occ+num_unique_words);
	}
}

void NaiveBayesClassifier::calculate_likelihood(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int number_unique_words, int number_documents, int number_labels, double* weights)
{
	printf("Calculating First parameter...\n");

	this->weights = weights;
	this->num_labels = number_labels;

	const int THREADS_Y = number_labels;
	const int THREADS_X = LIKELIHOOD_THREADS_X;
	const int BLOCKS_X = ceil(number_unique_words/(float)THREADS_X);
	const int BLOCKS_Y = ceil(number_labels/(float)THREADS_Y);
	const int FV_SIZE = (documents_indexes[number_documents-1] + documents_size[number_documents - 1]);

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int* d_fv, *d_doc_size, *d_doc_ind, *d_doc_label;
	double * d_fp, *h_fp;
	double * d_occurences;
	double*  d_weights;

	size_t pitch_fp,pitch_occ;

	h_fp = (double*)malloc(sizeof(double)*number_unique_words*number_labels);
	
	hipMallocPitch(&d_occurences,&pitch_occ,number_unique_words*sizeof(double),number_labels);
	hipMalloc((void**)&d_fv,FV_SIZE*sizeof(int));
	hipMalloc((void**)&d_doc_size,number_documents*sizeof(int));
	hipMalloc((void**)&d_doc_label,number_documents*sizeof(int));
	hipMalloc((void**)&d_doc_ind,number_documents*sizeof(int));
	hipMalloc((void**)&d_weights,number_documents*number_labels*sizeof(double));
	hipMallocPitch(&d_fp,&pitch_fp,number_unique_words*sizeof(double),number_labels);

	hipMemcpy(d_fv,feature_vectors,FV_SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_weights,weights,number_documents*number_labels*sizeof(double),hipMemcpyHostToDevice);

	hipMemcpy(d_doc_size,documents_size,number_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_doc_label,docs_labels,number_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_doc_ind,documents_indexes,number_documents*sizeof(int),hipMemcpyHostToDevice);

	calculate_occurences_kernel<<<blocks,threads>>>(d_fv,d_occurences,d_doc_size,d_doc_ind,d_doc_label,pitch_occ,number_documents,number_unique_words,number_labels,d_weights);
	calculate_likelihood_kernel<<<blocks,threads>>>(d_fp,d_occurences,pitch_occ,pitch_fp,number_unique_words,number_labels);

	hipMemcpy2D(h_fp,number_unique_words*sizeof(double),d_fp,pitch_fp,number_unique_words*sizeof(double),number_labels,hipMemcpyDeviceToHost);

	first_parameter = CudaStd::convert_1d_to_2d_float(h_fp,number_labels,number_unique_words);

	free(h_fp);

	hipFree(d_fv);
	hipFree(d_doc_label);
	hipFree(d_doc_size);
	hipFree(d_doc_ind);
	hipFree(d_fp);
	hipFree(d_occurences);
	hipFree(d_weights);
}

#else

void NaiveBayesClassifier::calculate_likelihood(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int number_unique_words, int number_documents, int number_labels, double* weights)
{
    printf("Calculating First parameter: \n");

	this->weights = weights;
	this->num_labels = number_labels;

	for(int i = 0;i < number_labels;i++)
	{
		int all_label_occ = calculate_all_words_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels,number_unique_words,number_documents,i);
		for(int j = 0; j < number_unique_words;j++)
		{
			first_parameter[i][j] = (calculate_single_word_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels ,j,number_documents,i) + 1)/
				(all_label_occ+number_unique_words);

			//if(j<30)
			//printf("%d ",calculate_single_word_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels ,j,number_documents,i));
		}
		//printf("\n");
	}
    
	//printf("\n");
}

double NaiveBayesClassifier::calculate_single_word_label_occurence(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int word_index, int number_documents, int label)
{
	double result = 0;

	for(int i = 0;i<number_documents;i++)
	{
		if(label == docs_labels[i])
		{
			for(int j = 0;j < documents_size[i];j++)
			{
				if(feature_vectors[documents_indexes[i] + j] == word_index)
				{
						result++;
				}
			}

		}
		else if(docs_labels[i] == -1 && (weights[i*num_labels + label] > EM_WEIGHTS_THRESHOLD) )
		{
			for(int j = 0;j < documents_size[i];j++)
			{
				if(feature_vectors[documents_indexes[i] + j] == word_index)
				{
						result += (weights[i*num_labels + label]*1);
				}
			}
		}
			
	}

	return result;

}

double NaiveBayesClassifier::calculate_all_words_label_occurence(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int number_unique_words, int number_documents, int label)
{
	double result = 0;

	for(int i = 0;i<number_unique_words;i++)
	{
		result += calculate_single_word_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels,i,number_documents,label);
	}

	return result;
}

#endif

#ifndef CUDA_CLASSIFY

int NaiveBayesClassifier::classify_unlabeled_document(int * unlabeled_feature_vector, int fv_size, int number_unique_words, int number_labels, double* prob_labels)
{
	double max_prob = N_INF;
	double min_prob = 1;
	int label = -1;
    
	for(int i = 0;i < number_labels;i++)
	{
		double prob = (double)NaiveBayesClassifier::prob_document_label(unlabeled_feature_vector, fv_size ,number_unique_words,i);

		if(prob_labels)
			prob_labels[i] = prob;

        if(prob > max_prob)
		{
			max_prob = prob;
			label = i;
		}

		if(prob < min_prob)
		{
			min_prob = prob;
		}
	}

	if(prob_labels)
	{
		for(int i = 0; i < number_labels;i++)
		{
			prob_labels[i] = (prob_labels[i] - min_prob)/(max_prob - min_prob);
		}
	}

	return label;
}

double NaiveBayesClassifier::prob_document_label(int * feature_vector, int fv_size, int number_unique_words, int label)
{
	double result = log10(second_parameter[label]);

	//printf("Label: %d Prior: %Lf \n",label,result);

	for(int i = 0;i<fv_size;i++)
	{
        if(feature_vector[i] == -1 || first_parameter[label][feature_vector[i]] == 0)
            continue;
		result = result + log10(first_parameter[label][feature_vector[i]]);
		//printf("result:  %f %d = %e \n",first_parameter[label][i], feature_vector[i] ,result);
	}

    //printf("Label: %d Final Prob: %Lf\n",label,result);
	return result;
}

#else


__device__ double prob_document_label(int * feature_vector,double* fp, double*sp, int fv_size, int number_unique_words, int label)
{
	
	double result = log10((double)sp[label]);
	//printf("Label: %d Prior: %Lf \n",label,result);

	for(int i = 0;i<fv_size;i++)
	{
        if(feature_vector[i] == -1 || (fp+label*number_unique_words)[feature_vector[i]] == 0)
            continue;
		result = result + log10((double)(fp+label*number_unique_words)[feature_vector[i]]);
		//printf("result:  %f %d = %e \n",first_parameter[label][i], feature_vector[i] ,result);
	}

    //printf("Label: %d Final Prob: %Lf\n",label,result);
	return result;
}

__device__ int classify_unlabeled_document(int * unlabeled_feature_vector,double* fp, double* sp, int fv_size, int number_unique_words, int number_labels, double* prob_labels)
{
	double max_prob = N_INF;
	double min_prob = 1;
	int label = -1;
    
	for(int i = 0;i < number_labels;i++)
	{
		double prob = (double)prob_document_label(unlabeled_feature_vector, fp, sp, fv_size ,number_unique_words,i);

		if(prob_labels)
			prob_labels[i] = prob;

        if(prob > max_prob)
		{
			max_prob = prob;
			label = i;
		}

		if(prob < min_prob)
		{
			min_prob = prob;
		}
	}

	if(prob_labels)
	{
		for(int i = 0; i < number_labels;i++)
		{
			prob_labels[i] = (prob_labels[i] - min_prob)/(max_prob - min_prob);
		}
	}

	return label;
}

__global__ void classify_unlabeled_docs_kernel(int * unlabeled_feature_vectors, int* doc_sizes, int* docs_indexes, int num_docs ,double* fp, double* sp, int number_unique_words, int number_labels, int* docs_labels, double* prob_labels)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;

	if(x < num_docs)
	{
		if(prob_labels)
			docs_labels[x] = classify_unlabeled_document(unlabeled_feature_vectors+docs_indexes[x],fp,sp,doc_sizes[x],number_unique_words,number_labels,&prob_labels[x*number_labels]);
		else 
			docs_labels[x] = classify_unlabeled_document(unlabeled_feature_vectors+docs_indexes[x],fp,sp,doc_sizes[x],number_unique_words,number_labels,0);
	}
}




#endif

void NaiveBayesClassifier::classify_unlabeled_documents(int* feature_vectors, int* docs_sizes, int* docs_indexes, int num_documents, int num_unique_words, int num_labels, int* labels, double* prob_labels)
{
#ifndef CUDA_CLASSIFY
	for(int i = 0;i < num_documents;i++)
	{
		if(prob_labels)
			labels[i] = classify_unlabeled_document(feature_vectors+docs_indexes[i],docs_sizes[i],num_unique_words,num_labels,&prob_labels[i*num_labels]);
		else labels[i] = classify_unlabeled_document(feature_vectors+docs_indexes[i],docs_sizes[i],num_unique_words,num_labels,0);
	}
#else
	const int THREADS_Y = CLASSIFY_THREADS_Y;
	const int THREADS_X = CLASSIFY_THREADS_X;
	const int BLOCKS_X = ceil(num_documents/(float)THREADS_X);
	const int BLOCKS_Y = ceil(1/(float)THREADS_Y);
	const int FV_SIZE = (docs_indexes[num_documents-1] + docs_sizes[num_documents - 1]);

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int* d_fv, *d_docs_sizes, *d_docs_indexes, *d_labels;
	double* d_fp, *d_sp, *d_prob_labels;
	double *h_fp;

	hipMalloc((void**)&d_fv,FV_SIZE*sizeof(int));
	hipMalloc((void**)&d_docs_sizes,num_documents*sizeof(int));
	hipMalloc((void**)&d_docs_indexes,num_documents*sizeof(int));
	hipMalloc((void**)&d_labels,num_documents*sizeof(int));
	hipMalloc((void**)&d_fp,num_unique_words*num_labels*sizeof(double));
	hipMalloc((void**)&d_sp,num_labels*sizeof(double));
	hipMalloc((void**)&d_prob_labels,num_documents*num_labels*sizeof(double));

	h_fp = CudaStd::convert_2d_to_1d_float(first_parameter,num_labels,num_unique_words);

	hipMemcpy(d_fv,feature_vectors,FV_SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_docs_sizes,docs_sizes,num_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_docs_indexes,docs_indexes,num_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_fp,h_fp,num_unique_words*num_labels*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_sp,second_parameter,num_labels*sizeof(double),hipMemcpyHostToDevice);

	classify_unlabeled_docs_kernel<<<blocks,threads>>>(d_fv,d_docs_sizes,d_docs_indexes,num_documents,d_fp,d_sp,num_unique_words,num_labels,d_labels,d_prob_labels);

	hipMemcpy(labels,d_labels,num_documents*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(prob_labels,d_prob_labels,num_documents*num_labels*sizeof(double),hipMemcpyDeviceToHost);

	free(h_fp);

	hipFree(d_fv);
	hipFree(d_docs_sizes);
	hipFree(d_docs_indexes);
	hipFree(d_labels);
	hipFree(d_fp);
	hipFree(d_sp);
	hipFree(d_prob_labels);

#endif
}

int NaiveBayesClassifier::calculate_label_occurance(int * documents_labels, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i < number_documents; i++)
	{
		if(documents_labels[i] == label)
			result++;
	}

	return result;
}

void NaiveBayesClassifier::calculate_prior(int* documents_labels, int number_documents, int number_labels)
{
    printf("Calculating Second Parameter: \n");
	int denom = number_documents+number_labels;
    
	for(int i = 0;i<number_labels;i++)
	{
		second_parameter[i] = (double)(1+calculate_label_occurance(documents_labels,number_documents,i))/
			(double)denom;
        //printf("%lf \n",second_parameter[i]);
	}
    
    //printf("\n");

}

double ** NaiveBayesClassifier::get_likelihood()
{
    return first_parameter;
}

double *  NaiveBayesClassifier::get_prior()
{
    return second_parameter;
}