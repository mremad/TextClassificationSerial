#include "hip/hip_runtime.h"
#include "NaiveBayesClassifier.h"

#define N_INF -1000000000

NaiveBayesClassifier::NaiveBayesClassifier(int number_labels,int number_unique_words)
{
	first_parameter = (long double**)malloc(sizeof(long double*)*number_labels);
	second_parameter = (long double*)malloc(sizeof(long double)*number_labels);

	for(int i = 0;i<number_labels;i++)
		first_parameter[i] = (long double*)malloc(sizeof(long double)*number_unique_words);


}

long double ** NaiveBayesClassifier::get_likelihood()
{
    return first_parameter;
}

long double *  NaiveBayesClassifier::get_prior()
{
    return second_parameter;
}

#ifdef CUDA_EMAD

__device__ int calculate_single_word_label_occurence(int * feature_vectors,size_t pitch, int word_index, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i<number_documents;i++)
	{
		if(label == ((int*)((char*)feature_vectors+i*pitch))[0])
			result += ((int*)((char*)feature_vectors+i*pitch))[word_index+1];
	}

	return result;

}

__global__ void calculate_occurences_kernel(int * d_fv, long double* d_fp ,size_t pitch_fv, size_t pitch_fp, int num_docs, int num_unique_words, int num_labels)
{
	
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y;
	
	if(x < num_unique_words && y < num_labels)
	{
		((long double*)((char*)d_fp+y*pitch_fp))[x] = (long double)calculate_single_word_label_occurence(d_fv,pitch_fv,x,num_docs,y);
	}
}

__global__ void calculate_likelihood_kernel(long double* d_fp,long double* d_occ, size_t pitch_occ, size_t pitch_fp, int num_unique_words, int num_labels)
{
	
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y;
	
	if(x < num_unique_words && y < num_labels)
	{
		int all_label_occ = 0;

		for(int i = 0;i < num_unique_words;i++)
			all_label_occ += ((long double*)((char*)d_occ+y*pitch_occ))[i];

		((long double*)((char*)d_fp+y*pitch_fp))[x] = (((long double*)((char*)d_fp+y*pitch_fp))[x] + 1)/
					(long double)(all_label_occ+num_unique_words);
	}
}

int* convert_2d_to_1d_int(int** src, int rows, int cols)
{
	int* dest = (int*)malloc(rows*cols*sizeof(int));

	for(int i = 0;i < rows;i++)
	{
		for(int j = 0;j < cols;j++)
		{
			dest[i*cols + j] = src[i][j];
		}

	}

	return dest;
}

long double** convert_1d_to_2d_float(long double* src, int rows, int cols)
{
	long double** dest = (long double**)malloc(rows*sizeof(long double*));

	for(int i = 0;i < rows;i++)
	{
		dest[i] = (long double*)malloc(cols*sizeof(long double));

		for(int j = 0;j < cols;j++)
		{
			dest[i][j] = src[i*cols + j];
		}

	}

	return dest;
}

void NaiveBayesClassifier::calculate_likelihood(int** feature_vectors,int number_unique_words, int number_documents, int number_labels)
{
	printf("Calculating First parameter...\n");
	const int THREADS_Y = number_labels;
	const int THREADS_X = 32;
	const int BLOCKS_X = ceil(number_unique_words/(float)THREADS_X);
	const int BLOCKS_Y = ceil(number_labels/(float)THREADS_Y);

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int* d_fv, *h_fv;
	long double * d_fp, *h_fp;
	long double * d_occurences;

	size_t pitch_fv,pitch_fp,pitch_occ;

	h_fv = convert_2d_to_1d_int(feature_vectors,number_documents,number_unique_words+1);
	h_fp = (long double*)malloc(sizeof(long double)*number_unique_words*number_labels);
	
	hipMallocPitch(&d_occurences,&pitch_occ,number_unique_words*sizeof(long double),number_labels);
	hipMallocPitch(&d_fv,&pitch_fv,(number_unique_words+1)*sizeof(int),number_documents);
	hipMallocPitch(&d_fp,&pitch_fp,number_unique_words*sizeof(long double),number_labels);


	hipMemcpy2D(d_fv,pitch_fv,h_fv,(number_unique_words+1)*sizeof(int),(number_unique_words+1)*sizeof(int),number_documents,hipMemcpyHostToDevice);
	calculate_occurences_kernel<<<blocks,threads>>>(d_fv,d_fp,pitch_fv,pitch_fp,number_documents,number_unique_words,number_labels);
	hipMemcpy2D(d_occurences,pitch_occ,d_fp,pitch_fp,number_unique_words*sizeof(long double),number_labels,hipMemcpyDeviceToDevice);
	calculate_likelihood_kernel<<<blocks,threads>>>(d_fp,d_occurences,pitch_occ,pitch_fp,number_unique_words,number_labels);

	hipMemcpy2D(h_fp,number_unique_words*sizeof(long double),d_fp,pitch_fp,number_unique_words*sizeof(long double),number_labels,hipMemcpyDeviceToHost);

	first_parameter = convert_1d_to_2d_float(h_fp,number_labels,number_unique_words);

	free(h_fp);
	free(h_fv);
	

}

#else

void NaiveBayesClassifier::calculate_likelihood(int ** feature_vectors,int number_unique_words, int number_documents, int number_labels)
{
    printf("Calculating First parameter: \n");
	for(int i = 0;i < number_labels;i++)
	{
		int all_label_occ = calculate_all_words_label_occurence(feature_vectors,number_unique_words,number_documents,i);
		for(int j = 0; j < number_unique_words;j++)
		{
			first_parameter[i][j] = (long double)(calculate_single_word_label_occurence(feature_vectors,j,number_documents,i) + 1)/
				(long double)(all_label_occ+number_unique_words);

			//printf("%f ",first_parameter[i][j]);
		}
		//printf("\n");
	}
    
    //printf("\n");
}

int NaiveBayesClassifier::calculate_single_word_label_occurence(int ** feature_vectors, int word_index, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i<number_documents;i++)
	{
		if(label == feature_vectors[i][0])
			result += feature_vectors[i][word_index+1];
	}

	return result;

}

int NaiveBayesClassifier::calculate_all_words_label_occurence(int ** feature_vectors, int number_unique_words, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i<number_unique_words;i++)
	{
		result += calculate_single_word_label_occurence(feature_vectors,i,number_documents,label);
	}

	return result;
}

#endif



void NaiveBayesClassifier::calculate_prior(int** feature_vectors, int number_documents, int number_labels)
{
    printf("Calculating Second Parameter: \n");
	int denom = number_documents+number_labels;
    
	for(int i = 0;i<number_labels;i++)
	{
		second_parameter[i] = (long double)(1+calculate_label_occurance(feature_vectors,number_documents,i))/
			(long double)denom;
        //printf("%lf \n",second_parameter[i]);
	}
    
    //printf("\n");

}

int NaiveBayesClassifier::calculate_label_occurance(int ** feature_vectors, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i < number_documents; i++)
	{
		if(feature_vectors[i][0] == label)
			result++;
	}

	return result;
}

int NaiveBayesClassifier::classify_unlabeled_document(int * unlabeled_feature_vector, int number_unique_words, int number_labels)
{
	long double max_prob = N_INF;
	int label = -1;
    
	for(int i = 0;i < number_labels;i++)
	{
		long double prob = (long double)NaiveBayesClassifier::prob_document_label(unlabeled_feature_vector,number_unique_words,i);

        if(prob > max_prob)
		{
			max_prob = prob;
			label = i;
		}
	}

	//unlabeled_feature_vector[0] = label;
	return label;
}

long double NaiveBayesClassifier::prob_document_all_labels(int * feature_vector, int number_unique_words, int number_labels)
{
	long double result = 0;

	for(int i = 0;i<number_labels;i++)
	{
		result += NaiveBayesClassifier::prob_document_label(feature_vector,number_unique_words,i);
	}

	return result;
}


long double NaiveBayesClassifier::prob_document_label(int * feature_vector, int number_unique_words, int label)
{
	long double result = log10(second_parameter[label]);

	//printf("Label: %d Prior: %Lf \n",label,result);

	for(int i = 0;i<number_unique_words;i++)
	{
        if(first_parameter[label][i] == 0)
            continue;
		result = result + (feature_vector[i+1]*log10(first_parameter[label][i]));
		//printf("result:  %f %d = %e \n",first_parameter[label][i], feature_vector[i] ,result);
	}

    //printf("Label: %d Final Prob: %Lf\n",label,result);
	return result;
}

