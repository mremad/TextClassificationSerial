#include "hip/hip_runtime.h"
#include "NaiveBayesClassifier.h"
#include "ConsolePrint.h"

#define N_INF -1000000000

NaiveBayesClassifier::NaiveBayesClassifier(int number_labels,int number_unique_words)
{
	first_parameter = (long double**)malloc(sizeof(long double*)*number_labels);
	second_parameter = (long double*)malloc(sizeof(long double)*number_labels);

	for(int i = 0;i<number_labels;i++)
		first_parameter[i] = (long double*)malloc(sizeof(long double)*number_unique_words);


}

#ifdef CUDA_LIKELIHOOD

__device__ int calculate_single_word_label_occurence(int * feature_vectors,int * documents_size, int * documents_indexes, int * docs_labels, int word_index, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i<number_documents;i++)
	{
		if(label == docs_labels[i])
		{
			for(int j = 0;j < documents_size[i];j++)
			{
				if(feature_vectors[documents_indexes[i] + j] == word_index)
					result++;
			}

		}
			
	}


	return result;

}

__global__ void calculate_occurences_kernel(int * d_fv, int* d_fp ,int * documents_size, int * documents_indexes, int * docs_labels, size_t pitch_fp, int num_docs, int num_unique_words, int num_labels)
{
	
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y;
	
	if(x < num_unique_words && y < num_labels)
	{
		((int*)((char*)d_fp+y*pitch_fp))[x] = calculate_single_word_label_occurence(d_fv,documents_size,documents_indexes,docs_labels,x,num_docs,y);
	}
}

__global__ void calculate_likelihood_kernel(long double* d_fp,int* d_occ, size_t pitch_occ, size_t pitch_fp, int num_unique_words, int num_labels)
{
	
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y;
	
	if(x < num_unique_words && y < num_labels)
	{
		int all_label_occ = 0;

		for(int i = 0;i < num_unique_words;i++)
			all_label_occ += ((int*)((char*)d_occ+y*pitch_occ))[i];

		((long double*)((char*)d_fp+y*pitch_fp))[x] = (((int*)((char*)d_occ+y*pitch_occ))[x] + 1)/
					(long double)(all_label_occ+num_unique_words);
	}
}

void NaiveBayesClassifier::calculate_likelihood(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int number_unique_words, int number_documents, int number_labels)
{
	printf("Calculating First parameter...\n");
	const int THREADS_Y = number_labels;
	const int THREADS_X = LIKELIHOOD_THREADS_X;
	const int BLOCKS_X = ceil(number_unique_words/(float)THREADS_X);
	const int BLOCKS_Y = ceil(number_labels/(float)THREADS_Y);
	const int FV_SIZE = (documents_indexes[number_documents-1] + documents_size[number_documents - 1]);

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int* d_fv, *d_doc_size, *d_doc_ind, *d_doc_label;
	long double * d_fp, *h_fp;
	int * d_occurences;

	size_t pitch_fp,pitch_occ;

	h_fp = (long double*)malloc(sizeof(long double)*number_unique_words*number_labels);
	
	hipMallocPitch(&d_occurences,&pitch_occ,number_unique_words*sizeof(int),number_labels);
	hipMalloc((void**)&d_fv,FV_SIZE*sizeof(int));
	hipMalloc((void**)&d_doc_size,number_documents*sizeof(int));
	hipMalloc((void**)&d_doc_label,number_documents*sizeof(int));
	hipMalloc((void**)&d_doc_ind,number_documents*sizeof(int));
	hipMallocPitch(&d_fp,&pitch_fp,number_unique_words*sizeof(long double),number_labels);

	hipMemcpy(d_fv,feature_vectors,FV_SIZE*sizeof(int),hipMemcpyHostToDevice);

	hipMemcpy(d_doc_size,documents_size,number_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_doc_label,docs_labels,number_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_doc_ind,documents_indexes,number_documents*sizeof(int),hipMemcpyHostToDevice);

	calculate_occurences_kernel<<<blocks,threads>>>(d_fv,d_occurences,d_doc_size,d_doc_ind,d_doc_label,pitch_occ,number_documents,number_unique_words,number_labels);
	calculate_likelihood_kernel<<<blocks,threads>>>(d_fp,d_occurences,pitch_occ,pitch_fp,number_unique_words,number_labels);

	hipMemcpy2D(h_fp,number_unique_words*sizeof(long double),d_fp,pitch_fp,number_unique_words*sizeof(long double),number_labels,hipMemcpyDeviceToHost);

	first_parameter = CudaStd::convert_1d_to_2d_float(h_fp,number_labels,number_unique_words);

	free(h_fp);

	hipFree(d_fv);
	hipFree(d_doc_label);
	hipFree(d_doc_size);
	hipFree(d_doc_ind);
	hipFree(d_fp);
	hipFree(d_occurences);
}

#else

void NaiveBayesClassifier::calculate_likelihood(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int number_unique_words, int number_documents, int number_labels)
{
    printf("Calculating First parameter: \n");
	for(int i = 0;i < number_labels;i++)
	{
		int all_label_occ = calculate_all_words_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels,number_unique_words,number_documents,i);
		for(int j = 0; j < number_unique_words;j++)
		{
			first_parameter[i][j] = (long double)(calculate_single_word_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels ,j,number_documents,i) + 1)/
				(long double)(all_label_occ+number_unique_words);

			if(j<30)
			printf("%d ",calculate_single_word_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels ,j,number_documents,i));
		}
		printf("\n");
	}
    
    printf("\n");
}

int NaiveBayesClassifier::calculate_single_word_label_occurence(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int word_index, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i<number_documents;i++)
	{
		if(label == docs_labels[i])
		{
			for(int j = 0;j < documents_size[i];j++)
			{
				if(feature_vectors[documents_indexes[i] + j] == word_index)
					result++;
			}

		}
			
	}

	return result;

}

int NaiveBayesClassifier::calculate_all_words_label_occurence(int * feature_vectors, int * documents_size, int * documents_indexes, int * docs_labels, int number_unique_words, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i<number_unique_words;i++)
	{
		result += calculate_single_word_label_occurence(feature_vectors, documents_size, documents_indexes, docs_labels,i,number_documents,label);
	}

	return result;
}

#endif

#ifndef CUDA_CLASSIFY

int NaiveBayesClassifier::classify_unlabeled_document(int * unlabeled_feature_vector, int fv_size, int number_unique_words, int number_labels)
{
	long double max_prob = N_INF;
	int label = -1;
    
	for(int i = 0;i < number_labels;i++)
	{
		long double prob = (long double)NaiveBayesClassifier::prob_document_label(unlabeled_feature_vector, fv_size ,number_unique_words,i);

        if(prob > max_prob)
		{
			max_prob = prob;
			label = i;
		}
	}

	return label;
}

long double NaiveBayesClassifier::prob_document_label(int * feature_vector, int fv_size, int number_unique_words, int label)
{
	long double result = log10(second_parameter[label]);

	//printf("Label: %d Prior: %Lf \n",label,result);

	for(int i = 0;i<fv_size;i++)
	{
        if(feature_vector[i] == -1 || first_parameter[label][feature_vector[i]] == 0)
            continue;
		result = result + log10(first_parameter[label][feature_vector[i]]);
		//printf("result:  %f %d = %e \n",first_parameter[label][i], feature_vector[i] ,result);
	}

    //printf("Label: %d Final Prob: %Lf\n",label,result);
	return result;
}

#else


__device__ long double prob_document_label(int * feature_vector,long double* fp, long double*sp, int fv_size, int number_unique_words, int label)
{
	
	long double result = log10((double)sp[label]);
	//printf("Label: %d Prior: %Lf \n",label,result);

	for(int i = 0;i<fv_size;i++)
	{
        if(feature_vector[i] == -1 || (fp+label*number_unique_words)[feature_vector[i]] == 0)
            continue;
		result = result + log10((double)(fp+label*number_unique_words)[feature_vector[i]]);
		//printf("result:  %f %d = %e \n",first_parameter[label][i], feature_vector[i] ,result);
	}

    //printf("Label: %d Final Prob: %Lf\n",label,result);
	return result;
}

__device__ int classify_unlabeled_document(int * unlabeled_feature_vector,long double* fp, long double* sp, int fv_size, int number_unique_words, int number_labels)
{
	long double max_prob = N_INF;
	int label = -1;
    
	for(int i = 0;i < number_labels;i++)
	{
		long double prob = (long double)prob_document_label(unlabeled_feature_vector, fp, sp, fv_size ,number_unique_words,i);

        if(prob > max_prob)
		{
			max_prob = prob;
			label = i;
		}
	}

	return label;
}

__global__ void classify_unlabeled_docs_kernel(int * unlabeled_feature_vectors, int* doc_sizes, int* docs_indexes, int num_docs ,long double* fp, long double* sp, int number_unique_words, int number_labels, int* docs_labels)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;

	if(x < num_docs)
	{
		docs_labels[x] = classify_unlabeled_document(unlabeled_feature_vectors+docs_indexes[x],fp,sp,doc_sizes[x],number_unique_words,number_labels);
	}
}




#endif

void NaiveBayesClassifier::classify_unlabeled_documents(int* feature_vectors, int* docs_sizes, int* docs_indexes, int num_documents, int num_unique_words, int num_labels, int* labels)
{
#ifndef CUDA_CLASSIFY
	for(int i = 0;i < num_documents;i++)
	{
		labels[i] = classify_unlabeled_document(feature_vectors+docs_indexes[i],docs_sizes[i],num_unique_words,num_labels);
	}
#else
	const int THREADS_Y = CLASSIFY_THREADS_Y;
	const int THREADS_X = CLASSIFY_THREADS_X;
	const int BLOCKS_X = ceil(num_documents/(float)THREADS_X);
	const int BLOCKS_Y = ceil(1/(float)THREADS_Y);
	const int FV_SIZE = (docs_indexes[num_documents-1] + docs_sizes[num_documents - 1]);

	dim3 blocks(BLOCKS_X,BLOCKS_Y);
	dim3 threads(THREADS_X,THREADS_Y);

	int* d_fv, *d_docs_sizes, *d_docs_indexes, *d_labels;
	long double* d_fp, *d_sp;
	long double *h_fp;

	hipMalloc((void**)&d_fv,FV_SIZE*sizeof(int));
	hipMalloc((void**)&d_docs_sizes,num_documents*sizeof(int));
	hipMalloc((void**)&d_docs_indexes,num_documents*sizeof(int));
	hipMalloc((void**)&d_labels,num_documents*sizeof(int));
	hipMalloc((void**)&d_fp,num_unique_words*num_labels*sizeof(long double));
	hipMalloc((void**)&d_sp,num_labels*sizeof(long double));

	h_fp = CudaStd::convert_2d_to_1d_float(first_parameter,num_labels,num_unique_words);

	hipMemcpy(d_fv,feature_vectors,FV_SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_docs_sizes,docs_sizes,num_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_docs_indexes,docs_indexes,num_documents*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_fp,h_fp,num_unique_words*num_labels*sizeof(long double),hipMemcpyHostToDevice);
	hipMemcpy(d_sp,second_parameter,num_labels*sizeof(long double),hipMemcpyHostToDevice);

	classify_unlabeled_docs_kernel<<<blocks,threads>>>(d_fv,d_docs_sizes,d_docs_indexes,num_documents,d_fp,d_sp,num_unique_words,num_labels,d_labels);

	hipMemcpy(labels,d_labels,num_documents*sizeof(int),hipMemcpyDeviceToHost);

	free(h_fp);

	hipFree(d_fv);
	hipFree(d_docs_sizes);
	hipFree(d_docs_indexes);
	hipFree(d_labels);
	hipFree(d_fp);
	hipFree(d_sp);

#endif
}

int NaiveBayesClassifier::calculate_label_occurance(int * documents_labels, int number_documents, int label)
{
	int result = 0;

	for(int i = 0;i < number_documents; i++)
	{
		if(documents_labels[i] == label)
			result++;
	}

	return result;
}

void NaiveBayesClassifier::calculate_prior(int* documents_labels, int number_documents, int number_labels)
{
    printf("Calculating Second Parameter: \n");
	int denom = number_documents+number_labels;
    
	for(int i = 0;i<number_labels;i++)
	{
		second_parameter[i] = (long double)(1+calculate_label_occurance(documents_labels,number_documents,i))/
			(long double)denom;
        //printf("%lf \n",second_parameter[i]);
	}
    
    //printf("\n");

}

long double ** NaiveBayesClassifier::get_likelihood()
{
    return first_parameter;
}

long double *  NaiveBayesClassifier::get_prior()
{
    return second_parameter;
}